#include "hip/hip_runtime.h"
#include "chamfermatch.h"
#include <stdio.h>
#include <algorithm>

// Chamfer version 2 should include CUB library
//#define ENABLE_CHAMFER_VERSION2

#ifdef ENABLE_CHAMFER_VERSION2
#define CUB_IGNORE_DEPRECATED_COMPILER
#define HIPCUB_STDERR
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <hipcub/hipcub.hpp>
///// Device-wise sorting
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <cub/../test/test_util.h>
#endif

__global__ void WarmUpGpu(){
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + index_x;
}

// NOTE: CUDA cannot use reference pointer as sender -> int& (X) int (O)
// Because the pointers must be copied to GPU memory first.
__global__ void DilateAndMoveMat(uchar* input_mat_data_8uc1, uchar* output_mat_data_8uc1,
                                 short src_w, short src_h,
                                 short kernel_w, short kernel_h,
                                 short resize_scale,
                                 short padding_w, short padding_h)
{
    ///
    ///
    ///
    int index_x = threadIdx.x + blockIdx.x * blockDim.x; // blockDim: thread_size of one block
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int index_z = threadIdx.z + blockIdx.z * blockDim.z;

    // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
    // printf("recomputed_kernel_w: %d\n", (int)src_w);
    // printf("recomputed_kernel_h: %d\n", (int)src_h);
    // Out of range
    if(index_x >= src_w + padding_w ||
       index_y >= src_h + padding_h ){
        // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
        // printf("recomputed_kernel_w: %d\n", (int)src_w);
        // printf("recomputed_kernel_h: %d\n", (int)src_h);
        return;
    }
    // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
    // printf("----------------- DilateAndMoveMat -------------------\n");

    // index in the shared memory
    int current_input_img_index = (index_x - padding_w) + (index_y - padding_h) * src_w;

    // Target output data information
    int target_width = src_w * resize_scale + (padding_w * 2);
    int target_height = src_h * resize_scale + (padding_h * 2);
    int target_index_x = index_x;// * resize_scale + padding_w;
    int target_index_y = index_y;// * resize_scale + padding_h;
    int target_index = target_index_x + target_index_y * target_width;
    output_mat_data_8uc1[target_index] = 0;

    short recomputed_kernel_w = (src_w - index_x + padding_w < kernel_w) ? src_w - index_x + padding_w : kernel_w;
    short recomputed_kernel_h = (src_h - index_y + padding_h < kernel_h) ? src_h - index_y + padding_h : kernel_h;
    short recomputed_start_w = (index_x - padding_w < 0) ? padding_w - index_x : 0;
    short recomputed_start_h = (index_y - padding_h < 0) ? padding_h - index_y : 0;
    // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
    // printf("----------------- DilateAndMoveMat -------------------\n");
    // printf("recomputed_kernel_w: %d\n", (int)recomputed_kernel_w);
    // printf("recomputed_kernel_h: %d\n", (int)recomputed_kernel_h);

    int index_searching = 0;
    bool found = false;
    for (int i = recomputed_start_h; i < recomputed_kernel_h; ++i) {
        for (int j = recomputed_start_w; j < recomputed_kernel_w; ++j) {
            index_searching = current_input_img_index + j + (i * src_w);
            // If there is any contours in the area, draw a point
            if(input_mat_data_8uc1[index_searching] > 127){
                output_mat_data_8uc1[target_index] = 255;
                // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
                // printf("----------------- DilateAndMoveMat: a point is drawn -------------------\n");
                found = true;
                break;
            }
        }
        if(found) break;
    }
}

__global__ void ReduceContourMat(uchar* input_mat_data_8uc1, uchar* output_mat_data_8uc1,
                                 short src_w, short src_h,
                                 short skip_kernel_size)
{
    ///
    ///
    ///
    int index_x = threadIdx.x + blockIdx.x * blockDim.x; // blockDim: thread_size of one block
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int index_z = threadIdx.z + blockIdx.z * blockDim.z;


    // Target output data information
    int target_index_x = index_x * skip_kernel_size;
    int target_index_y = index_y * skip_kernel_size;
    int target_index = target_index_x + target_index_y * src_w;

    // Out of range
    if(target_index_x >= src_w ||
       target_index_y >= src_h ){
        return;
    }

    int index_searching = 0;
    bool found = false;
    for (int i = 0; i < skip_kernel_size; ++i) {
        if(target_index_y + i >= src_h) continue;
        for (int j = 0; j < skip_kernel_size; ++j) {
            if(target_index_x + j >= src_w) continue;
            index_searching = target_index + j + (i * src_w);

            if(found == false){
                if(input_mat_data_8uc1[index_searching] > 127){
                    output_mat_data_8uc1[index_searching] = 255;
                    found = true;
                }
                else{
                    output_mat_data_8uc1[index_searching] = 0;
                }
            }
            else{
                output_mat_data_8uc1[index_searching] = 0;
            }
        }
    }
}


__global__ void ChamferMatch(float* flat_source_dist_transfrom,
                             short* source_candidate_col, short* source_candidate_row,
                             int source_width, int source_height,
                             int num_templates,
                             short* flat_template_non_zero_col, short* flat_template_non_zero_row, int* template_num_non_zero,
                             float* flat_template_weight, short use_weight,
                             int length_source_candidate, int source_num_candidate_width, int source_num_candidate_height,
                             float* flat_score_map, int* flat_score_map_index)
{
    ///
    /// flat_template_non_zero_col = [scaling x rotation][num_non_zeros]
    ///
    /// num_templates: angles + scales
    ///
    /// (at the first stage) source_candidate_ is obtained by findNonZero function
    /// source_candidate_ is 1-d pointer. In order to use x and y threads at the same time, they should be separated to 2-d
    /// size of source_candidate_ =  max(index_x) * max(index_y)
    /// For example) size is 25 -> can be 5 x 5
    /// index_x: index of source_candidate_
    /// index_y: index of source_candidate_
    ///
    ///
    /// x: column, y: row, z: ?
    int index_x = threadIdx.x + blockIdx.x * blockDim.x; // blockDim: thread_size of one block
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int index_z = threadIdx.z + blockIdx.z * blockDim.z;

    // Out of range
    if(index_x >= source_num_candidate_width ||
       index_y >= source_num_candidate_height ||
       index_x + index_y * source_num_candidate_width >= length_source_candidate){
        // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
        // printf("Out of range - x: %d, y: %d\n", index_x, index_y);
        return;
    }

    // Get the thread index
    int candidate_index = index_x + index_y * source_num_candidate_width;
    // Get the target position on the source image
    int current_index = source_candidate_col[candidate_index] + source_candidate_row[candidate_index] * source_width;

    int template_index = 0;
    float tmp_val;

    for(short i = 0; i < num_templates; i ++){
        flat_score_map[candidate_index + i*length_source_candidate] = 0;
        flat_score_map_index[candidate_index + i*length_source_candidate] = candidate_index + i*length_source_candidate;
        for(short j= 0; j < template_num_non_zero[i]; j ++){
            int non_zero_index = flat_template_non_zero_col[template_index] + flat_template_non_zero_row[template_index] * source_width;
            // Jang 20220330
            if(use_weight == 1){
                float tmp_val_0 = flat_source_dist_transfrom[current_index + non_zero_index];
                float tmp_val_1 = flat_template_weight[template_index];
                tmp_val = tmp_val_0 * tmp_val_1;
                flat_score_map[candidate_index + i*length_source_candidate] += tmp_val;
                // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
                // if(flat_template_weight[template_index] > 1 || flat_template_weight[template_index] <= 0){
                //     printf("HERE:::::: flat_template_weight: %f \nflat_source_dist_transfrom: %f\n", flat_template_weight[template_index], flat_source_dist_transfrom[current_index + non_zero_index]);
                //     printf("HERE:::::: flat_template_weight: %f \n", flat_template_weight[template_index]);
                // }
            }
            else{
                flat_score_map[candidate_index + i*length_source_candidate] += flat_source_dist_transfrom[current_index + non_zero_index];
            }

            template_index++;
        }
        // average score
        flat_score_map[candidate_index + i*length_source_candidate] /= (float)template_num_non_zero[i];

    }
}


__global__ void ChamferMatch2(float* flat_source_dist_transfrom,
                             short* source_candidate_col, short* source_candidate_row,
                             int source_width, int source_height,
                             int num_templates,
                             short* flat_template_non_zero_col, short* flat_template_non_zero_row, int* template_num_non_zero,
                             float* flat_template_weight, short use_weight,
                             int length_source_candidate, int source_num_candidate_width, int source_num_candidate_height,
                             float* flat_score_map, int* flat_score_map_index)
{
    ///
    /// flat_template_non_zero_col = [scaling x rotation][num_non_zeros]
    ///
    /// num_templates: angles + scales
    ///
    /// (at the first stage) source_candidate_ is obtained by findNonZero function
    /// source_candidate_ is 1-d pointer. In order to use x and y threads at the same time, they should be separated to 2-d
    /// size of source_candidate_ =  max(index_x) * max(index_y)
    /// For example) size is 25 -> can be 5 x 5
    /// index_x: index of source_candidate_
    /// index_y: index of source_candidate_
    ///
    /// source_width: width of the image; including expanding(padding) option.
    ///
    /// x: column, y: row, z: ?
    ///
    ///
    int index_x = threadIdx.x + blockIdx.x * blockDim.x; // blockDim: thread_size of one block
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int index_z = threadIdx.z + blockIdx.z * blockDim.z;


    // Out of range
    if(index_x >= source_num_candidate_width ||
       index_y >= source_num_candidate_height ||
       index_x + index_y * source_num_candidate_width >= length_source_candidate){
//        printf("Out of range - x: %d, y: %d\n", index_x, index_y);
        return;
    }

    // Get the thread index
    int candidate_index = index_x + index_y * source_num_candidate_width;
    // Get the target position on the source image
    int current_index = source_candidate_col[candidate_index] + source_candidate_row[candidate_index] * source_width;

    int template_index = 0;
    int error_count_template_weight_0 = 0;
    int error_count_template_weight_1 = 0;
    float tmp_val;

    for(int i = 0; i < num_templates; i ++){
        flat_score_map[candidate_index + i*length_source_candidate] = 0;
        flat_score_map_index[candidate_index + i*length_source_candidate] = candidate_index + i*length_source_candidate;
        for(int j= 0; j < template_num_non_zero[i]; j ++){
            int non_zero_index = flat_template_non_zero_col[template_index] + flat_template_non_zero_row[template_index] * source_width;
            if(use_weight == 1){
                float tmp_val_0 = flat_source_dist_transfrom[current_index + non_zero_index];
                float tmp_val_1 = flat_template_weight[template_index];
                tmp_val = tmp_val_0 * tmp_val_1;
                flat_score_map[candidate_index + i*length_source_candidate] += tmp_val;
                // [OPTIMIZED] Commented out debug printfs and error counters for industrial code cleanliness
                if(flat_template_weight[template_index] > 1){
                    // error_count_template_weight_0++;
                    // printf("HERE:::::: flat_template_weight: %f \n", flat_template_weight[template_index]);
                }
                if(flat_template_weight[template_index] <= 0){
                    // error_count_template_weight_1++;
                    // printf("HERE:::::: flat_template_weight: %f \nflat_source_dist_transfrom: %f\n", flat_template_weight[template_index], flat_source_dist_transfrom[current_index + non_zero_index]);
                }
            }
            else{
                flat_score_map[candidate_index + i*length_source_candidate] += flat_source_dist_transfrom[current_index + non_zero_index];
            }
            template_index++;
        }
        // average score
        flat_score_map[candidate_index + i*length_source_candidate] /= (float)template_num_non_zero[i];

    }
    // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
    // printf("HERE:::::: error_count_template_weight_0: %d \n", error_count_template_weight_0);
    // printf("HERE:::::: error_count_template_weight_1: %d \n", error_count_template_weight_1);
}

__global__ void TestAccessMem(float* shared_mem){
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;

    if(index_x > 100) return;
    for(int i = 0; i < 100; i ++){
        shared_mem[i + index_x] = 100;
    }


}

__global__ void SortChamferScores(float* score_candidates, int* score_candidates_index,
                             float* result_score_candidates, int* result_score_candidates_index,
                             int num_scores,
                             int target_num_scores, int divider,
                             float threshold_score, float max_score){
    ///
    /// num_scores: the number of score candidates
    /// target_num_scores: the number of result scores in one thread
    /// divider: divide num_scores by target_num_scores (the number of sorting blocks)
    ///
    /// (num_scores) / (divider) = a block size of the sorting
    /// (target_num_scores) * (num_scores) / (divider) = the result size
    ///
    /// threshold_score: threshold of scores
    /// max_score: if the score is higher than threshold_score, set the score as max_score
    ///


    int index_x = threadIdx.x + blockIdx.x * blockDim.x; // index_x: index of current sorting block
//    int index_y = threadIdx.y + blockIdx.y * blockDim.y;

    ///
    /// if num_scores = 150000
    /// sorting_block_size = 150000 / 150 = 1000
    ///
    int sorting_block_size = (num_scores / divider);    // At the last block, its size can be smaller 'sorting_block_size'
    if(num_scores % divider > 0) sorting_block_size++;  // ceil function

    // Out of range
    if(index_x >= divider){
        return;
    }

    int current_block_index = index_x * sorting_block_size;
    int current_result_index = index_x * target_num_scores;
    // init scores
    for(int i = 0; i < target_num_scores; i ++){
        result_score_candidates[i + current_result_index] = max_score;
    }


    for(int i = 0; i < sorting_block_size; i ++){
        if((i + current_block_index) >=  num_scores){
            break;
        }

        if(score_candidates[i + current_block_index] > threshold_score){
            continue;
        }

        if(score_candidates[i + current_block_index] == 0){
            continue;
        }


        for(int j = 0; j < target_num_scores; j ++){
            if(result_score_candidates[j + current_result_index] > score_candidates[i + current_block_index] ){
                // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
                // if(score_candidates[i + current_block_index] == 0){
                //     printf("HERE:::::: threadIdx.x: %d, threadIdx.y: %d \n ", threadIdx.x, threadIdx.y);
                // }
                // Make a space for new value
                for(int k = target_num_scores - 1; k > j; k--){
                    result_score_candidates[k + current_result_index] = result_score_candidates[k - 1 + current_result_index];
                    result_score_candidates_index[k + current_result_index] = result_score_candidates_index[k - 1 + current_result_index];
                }
                // Insert the new value
                result_score_candidates[j + current_result_index] = score_candidates[i + current_block_index] ;
                result_score_candidates_index[j + current_result_index] = score_candidates_index[i + current_block_index];

                // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
                // if(target_num_scores == 10){
                //     printf("HERE:::::: %f, %d, %d\n", result_score_candidates[j + current_result_index], i, j);
                // }
                // if(result_score_candidates[j + current_result_index] > 10000000){
                //     printf("HERE:::::: %f, %d, %d\n", result_score_candidates[j + current_result_index], i, j);
                //     // printf(" %f, %d, %d, %d\n", result_score_candidates[j + current_result_index+ 1], index_x, divider, num_scores);
                // }
                break;
            }
        }
    }

}


void CudaChamfer::SortScores(float* cuda_flat_score_map, int* cuda_flat_score_map_index,
                float* score_candidates_result, int* score_candidates_result_index,
                int num_scores, int target_num_scores, int steps,
                bool is_fine_detection){
    /// Input
    /// cuda_flat_score_map: scores
    /// cuda_flat_score_map_index: index of scores
    /// num_scores: the number of scores(length of array)
    /// target_num_scores: the target score lists. ex) Top 10 scores -> put 10
    /// steps: how many times sorting repeats.
    /// is_fine_detection: original matching(true) or downsized matching (false)
    ///
    /// Output
    /// score_candidates_result: the result scores of sorting
    /// score_candidates_result_index: ..



    // [OPTIMIZED] Add runtime parameter validation and error handling
    if (num_scores <= 1 || target_num_scores <= 0 || num_scores < target_num_scores || steps <= 0 || num_scores <= pow(10, steps) * target_num_scores) {
        fprintf(stderr, "[ERROR] Invalid parameters in SortScores.\n");
        return;
    }

    float threshold_score;
    float max_score;
    if(is_fine_detection){
        threshold_score = threshold_score_orig;
        max_score = max_score_orig;
    }
    else{
        threshold_score = threshold_score_py;
        max_score = max_score_py;
    }

    int num_scores_digits;
    int target_num_scores_digits = 0;
    int step_size;
    for(int i = 0; i < 1000; i++){
        double pow_val = pow(10, i);
        if(pow_val >= target_num_scores
           && target_num_scores_digits == 0){
            target_num_scores_digits = i - 1;
        }
        if(pow_val >= num_scores){
            num_scores_digits = i - 1;

            step_size = (float)(i - 1 - target_num_scores_digits) / (float)steps;
            break;
        }
    }
#ifdef DEBUG_MODE
    lw << "num_scores: " << num_scores << lw.endl;
    lw << "num_scores_digits: " << num_scores_digits << lw.endl;
    lw << "step_size: " << step_size << lw.endl;
#endif
    int num_score_blocks;
    int target_num_scores_for_each_step;
    int block_size = 32;    // max: 32

    dim3 threads_in_a_block;
    dim3 num_blocks;;

    float* cuda_score_candidates = cuda_flat_score_map;
    int* cuda_score_candidates_index = cuda_flat_score_map_index;

    int threads_digits;
    int num_threads;
    int target_num_per_thread;
    for(int i = 0; i < steps; i++){
        target_num_scores_for_each_step = num_scores / pow(10, step_size);    // 10,000,000 / 100 = 100,000 = divider * target_num_scores
        //  600,000 / 100 = 6,000 -> 1000  1st
        // 1000 / 100 = 10  2nd
//        lw << "target_num_scores_for_each_step: " << target_num_scores_for_each_step << lw.endl;
        if(target_num_scores_for_each_step > pow(10, step_size) * pow(10, target_num_scores_digits) ){
            num_scores_digits = num_scores_digits - step_size;  // 7 - 3
//            threads_digits = ceil((float)num_scores_digits / 2.0);       // 5 / 2 = 3
//            num_threads = pow(10, threads_digits);              // 1000
//            target_num_per_thread = pow(10, num_scores_digits - threads_digits);    // 10^(5-3) = 100

            target_num_per_thread = 10;    // 10^(5-3) = 100
            threads_digits = num_scores_digits - 1;       // 5 / 2 = 3
            num_threads = pow(10, threads_digits);              // 1000

        }
        else{
            num_threads = 1;
            target_num_per_thread = target_num_scores;
        }
        target_num_scores_for_each_step = target_num_per_thread * num_threads;
#ifdef DEBUG_MODE
        lw << "target_num_scores_for_each_step: " << target_num_scores_for_each_step << lw.endl;
        lw << "target_num_per_thread: " << target_num_per_thread << lw.endl;
        lw << "num_scores_digits: " << num_scores_digits << lw.endl;
        lw << "num_threads: " << num_threads << lw.endl;
#endif

        float* cuda_score_candidates_result = NULL;
        int* cuda_score_candidates_index_result = NULL;
        if (hipMalloc(&cuda_score_candidates_result, target_num_scores_for_each_step * sizeof(float)) != hipSuccess) {
            fprintf(stderr, "[ERROR] hipMalloc failed for cuda_score_candidates_result.\n");
            return;
        }
        if (hipMalloc(&cuda_score_candidates_index_result, target_num_scores_for_each_step * sizeof(int)) != hipSuccess) {
            fprintf(stderr, "[ERROR] hipMalloc failed for cuda_score_candidates_index_result.\n");
            hipFree(cuda_score_candidates_result);
            return;
        }

        block_size = num_scores / num_threads;
        if(block_size > 32) block_size = 32;
        if(block_size < 1) block_size = 1;
        threads_in_a_block = dim3(block_size);
        num_score_blocks = (int)((num_threads + block_size - 1) / block_size);
        num_blocks = dim3(num_score_blocks);

//        hipDeviceSynchronize();
        // [OPTIMIZED] Add CUDA error checking for kernel launch
        SortChamferScores<<<num_blocks, threads_in_a_block>>>(cuda_score_candidates, cuda_score_candidates_index,
                                                             cuda_score_candidates_result, cuda_score_candidates_index_result,
                                                             num_scores,
                                                             target_num_per_thread, num_threads,
                                                             threshold_score, max_score);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "[ERROR] SortChamferScores kernel launch failed: %s\n", hipGetErrorString(err));
            hipFree(cuda_score_candidates_result);
            hipFree(cuda_score_candidates_index_result);
            return;
        }
//        hipDeviceSynchronize();
        FreeCudaMemory(&cuda_score_candidates);
        FreeCudaMemory(&cuda_score_candidates_index);
        num_scores = target_num_scores_for_each_step;
        cuda_score_candidates = cuda_score_candidates_result;
        cuda_score_candidates_index = cuda_score_candidates_index_result;
    }
    cuda_flat_score_map = NULL;
    cuda_flat_score_map_index = NULL;

    // Copy to CPU
    // [OPTIMIZED] Add CUDA error checking for memory copy
    if (hipMemcpy(score_candidates_result, cuda_score_candidates, num_scores * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "[ERROR] hipMemcpy failed for score_candidates_result.\n");
    }
    if (hipMemcpy(score_candidates_result_index, cuda_score_candidates_index, num_scores * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "[ERROR] hipMemcpy failed for score_candidates_result_index.\n");
    }


    FreeCudaMemory(&cuda_score_candidates);
    FreeCudaMemory(&cuda_score_candidates_index);
}

void CudaChamfer::SortScores(float* cuda_flat_score_map, int* cuda_flat_score_map_index,
                float* score_candidates_result, int* score_candidates_result_index,
                int num_scores_copy, int steps,
                std::vector<int>& num_score_list, std::vector<int>& divider_list,
                bool is_fine_detection){
    /// Input
    /// cuda_flat_score_map: scores
    /// cuda_flat_score_map_index: index of scores
    /// num_scores_copy: the number of scores(length of array)
    /// steps: how many times sorting repeats.
    /// num_score_list
    /// divider_list
    /// is_fine_detection: original matching(true) or downsized matching (false)
    ///
    /// Output
    /// score_candidates_result: the result scores of sorting
    /// score_candidates_result_index: ..


    // [OPTIMIZED] Add runtime parameter validation and error handling
    if (steps <= 0 || num_score_list.size() != steps || divider_list.size() != steps || num_scores_copy <= 1) {
        fprintf(stderr, "[ERROR] Invalid parameters in SortScores (vector version).\n");
        return;
    }

    int num_threads;
    int target_num_per_thread;

    int num_score_blocks;
    int target_num_scores_for_each_step;
    int block_size = 32;    // max: 32
    int available_cores = 384; // Jetson Xavier NX

    dim3 threads_in_a_block;
    dim3 num_blocks;;

    float* cuda_score_candidates = cuda_flat_score_map;
    int* cuda_score_candidates_index = cuda_flat_score_map_index;


    for(int i = 0; i < steps; i++){
        target_num_per_thread = num_score_list.at(i);
        num_threads = divider_list.at(i);
        if(i > 0){
            assert(target_num_scores_for_each_step > target_num_per_thread * num_threads);
        }
        target_num_scores_for_each_step = target_num_per_thread * num_threads;
    }


    int num_scores = num_scores_copy;

    float threshold_score;
    float max_score;
    if(is_fine_detection){
        threshold_score = threshold_score_orig;
        max_score = max_score_orig;
    }
    else{
        threshold_score = threshold_score_py;
        max_score = max_score_py;
    }




    for(int i = 0; i < steps; i++){
        target_num_per_thread = num_score_list.at(i);
        num_threads = divider_list.at(i);
        target_num_scores_for_each_step = target_num_per_thread * num_threads;
#ifdef DEBUG_MODE
        lw << "target_num_scores_for_each_step: " << target_num_scores_for_each_step << lw.endl;
        lw << "target_num_per_thread: " << target_num_per_thread << lw.endl;
        lw << "num_threads: " << num_threads << lw.endl;
#endif

        float* cuda_score_candidates_result = nullptr;
        int* cuda_score_candidates_index_result = nullptr;
        if (hipMalloc(&cuda_score_candidates_result, target_num_scores_for_each_step * sizeof(float)) != hipSuccess) {
            fprintf(stderr, "[ERROR] hipMalloc failed for cuda_score_candidates_result.\n");
            return;
        }
        if (hipMalloc(&cuda_score_candidates_index_result, target_num_scores_for_each_step * sizeof(int)) != hipSuccess) {
            fprintf(stderr, "[ERROR] hipMalloc failed for cuda_score_candidates_index_result.\n");
            hipFree(cuda_score_candidates_result);
            return;
        }


        block_size = num_scores / available_cores + 1;
        if(block_size > 32) block_size = 32;
        if(block_size < 1) block_size = 1;
        threads_in_a_block = dim3(block_size);
        num_score_blocks = (int)((num_threads + block_size - 1) / block_size);
        num_blocks = dim3(num_score_blocks);

//        hipDeviceSynchronize();
        // [OPTIMIZED] Add CUDA error checking for kernel launch
        SortChamferScores<<<num_blocks, threads_in_a_block>>>(cuda_score_candidates, cuda_score_candidates_index,
                                                             cuda_score_candidates_result, cuda_score_candidates_index_result,
                                                             num_scores,
                                                             target_num_per_thread, num_threads,
                                                             threshold_score, max_score);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "[ERROR] SortChamferScores kernel launch failed: %s\n", hipGetErrorString(err));
            hipFree(cuda_score_candidates_result);
            hipFree(cuda_score_candidates_index_result);
            return;
        }
//        hipDeviceSynchronize();
        FreeCudaMemory(&cuda_score_candidates);
        FreeCudaMemory(&cuda_score_candidates_index);
        num_scores = target_num_scores_for_each_step;

        cuda_score_candidates = cuda_score_candidates_result;
        cuda_score_candidates_index = cuda_score_candidates_index_result;
    }
//    cuda_flat_score_map = nullptr;
//    cuda_flat_score_map_index = nullptr;

    // Copy to CPU
    // [OPTIMIZED] Add CUDA error checking for memory copy
    if (hipMemcpy(score_candidates_result, cuda_score_candidates, num_scores * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "[ERROR] hipMemcpy failed for score_candidates_result.\n");
    }
    if (hipMemcpy(score_candidates_result_index, cuda_score_candidates_index, num_scores * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "[ERROR] hipMemcpy failed for score_candidates_result_index.\n");
    }

    FreeCudaMemory(&cuda_score_candidates);
    FreeCudaMemory(&cuda_score_candidates_index);
}

#ifdef ENABLE_CHAMFER_VERSION2
using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory
int device_wise_sort(float* input_value, int* input_index, int input_size, float* output_value, int* output_index, int target_size)
{
//    printf("hipcub::DeviceRadixSort::SortPairs() %d items (%d-byte keys %d-byte values)\n",
//        input_size, int(sizeof(float)), int(sizeof(int)));
//    fflush(stdout);

    // Allocate device arrays
    float* cu_output_val = nullptr;
    int* cu_output_index = nullptr;
    hipMalloc(&cu_output_val, input_size * sizeof(float));
    hipMalloc(&cu_output_index, input_size * sizeof(int));
    DoubleBuffer<float> d_keys(input_value, cu_output_val);
    DoubleBuffer<int>   d_values(input_index, cu_output_index);
    // Allocate temporary storage
    size_t  temp_storage_bytes  = 0;
    void    *d_temp_storage     = NULL;
    DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, input_size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run
    DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, input_size);


    float   *h_keys_result              = new float[input_size];
    int     *h_values_result           = new int[input_size];
    hipMemcpy(h_keys_result, d_keys.Current(), sizeof(float) * input_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_values_result, d_values.Current(), sizeof(int) * input_size, hipMemcpyDeviceToHost);
    // [OPTIMIZED] Commented out debug printfs for industrial code cleanliness
    // if(dm::log){
    //     for (int i = 0; i < 20 ; ++i) {
    //         printf("%d: %d, %f\n", i, h_values_result[i], h_keys_result[i]);
    //     }
    // }

    for (int i = 0; i < target_size; ++i) {
        output_value[i] = h_keys_result[i];
        output_index[i] = h_values_result[i];
    }
    if (h_values_result) delete[] h_values_result;
    if (h_keys_result) delete[] h_keys_result;

    // [OPTIMIZED] Commented out debug printfs and unused code for industrial code cleanliness
    // if(dm::log){
    //     printf("---------------------------------\n");
    //     h_keys_result              = new float[input_size];
    //     h_values_result           = new int[input_size];
    //     hipMemcpy(h_keys_result, d_keys.d_buffers[0], sizeof(float) * input_size, hipMemcpyDeviceToHost);
    //     hipMemcpy(h_values_result, d_values.d_buffers[0], sizeof(int) * input_size, hipMemcpyDeviceToHost);
    //     // Print results
    //     for (int i = 0; i < 30 ; ++i) {
    //         printf("%d: %d, %f\n", i, h_values_result[i], h_keys_result[i]);
    //     }
    //     printf("\n");
    //     // Cleanup
    //     if (h_values_result) delete[] h_values_result;
    //     if (h_keys_result) delete[] h_keys_result;
    // }
    // if (d_keys.d_buffers[0]) hipFree(d_keys.d_buffers[0]);
    // if (d_values.d_buffers[0]) hipFree(d_values.d_buffers[0]);
    if (d_temp_storage) hipFree(d_temp_storage);
    if (cu_output_val) hipFree(cu_output_val);
    if (cu_output_index) hipFree(cu_output_index);
    return 0;
}
#endif

float CudaChamfer::ChamferInference(cv::Mat source_img, bool use_weight){
    // Jang 20220713
    const std::string pyr_path = data_path + "/pyr_source/";
    const std::string orig_path = data_path + "/orig_source/";
    const std::string result_path = data_path + "/results/";
    if(fs::exists(orig_path)){
       fs::remove_all(orig_path);
    }
    if(fs::exists(pyr_path)){
       fs::remove_all(pyr_path);
    }
    if(fs::exists(result_path)){
       fs::remove_all(result_path);
    }
    fs::create_directories(pyr_path);
    fs::create_directories(orig_path);
    fs::create_directories(result_path);

    // Remove result data
    if(fs::exists(data_path + result_score_path))
        fs::remove(data_path + result_score_path);

    // Jang 20220711
    // Should be modified
    lw << "[DEBUG] chamfer_method_: " << chamfer_method_
       << ", File " <<  __FILE__ <<  ", line " << __LINE__
       << ", in " << __func__ << lw.endl;

//    use_weight_ = 1;
    fail_msg = "";

    if(trained_template.non_zero_area_templates.size() < 1 ||
        trained_template_pyr.non_zero_area_templates.size() < 1){
        lw << "[FAIL] No template is detected"
           << ", File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;
        t_execute = 0;
        return 0;
    }
    if(source_img.empty()){
        lw << "[FAIL] No source image is detected"
           << ", File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;
        t_execute = 0;
        return 0;
    }

    if(dm::log){
        hipDeviceSynchronize();
    }
    struct timespec start_all;//clock_t start_all = clock();
    clock_gettime(CLOCK_MONOTONIC, &start_all);
    struct timespec tp_start, tp_end;
    double host_elapsed;
    clock_gettime(CLOCK_MONOTONIC, &tp_start);
    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }

    orig_source_img_mat_ = source_img.clone();

    if(dm::log){
    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
       << ", in " << __func__ << lw.endl;
    }
    // Paras. setting #FIXME
    float angle_interval_fine_align = 1.0;
    float angle_interval_coarse_align = 5.0;
    float angle_min = -45.0;
    float angle_max = 45.0;
    int num_angles;
    int num_pyramid = 1;
    rslt_match.clear();

    std::string params_path = this->params_path_ + parameter_matching_file;
    if (!fio::exists(params_path)){
        if(dm::log){
            std::cerr << "File isn't opened." << std::endl;
        }
        CreateParamsIniFile(params_path);
    }else {
        // FileIO. 20220419. Jimmy. #fio14
        fio::FileIO inim(params_path, fio::FileioFormat::INI);
        inim.IniSetSection("ParameterChamfer");
        angle_min = inim.IniReadtoFloat("angle_min");
        angle_max = inim.IniReadtoFloat("angle_max");
        angle_interval_coarse_align = inim.IniReadtoFloat("angle_coarse_interval");
        angle_interval_fine_align = inim.IniReadtoFloat("angle_fine_interval");

        inim.close();
    }
    num_angles = (angle_max - angle_min ) / angle_interval_fine_align;


    assert(angle_min < angle_max);
    assert(angle_min >= -180);
    assert(angle_max <= 180);
    assert(angle_interval_fine_align <= angle_interval_coarse_align);
    assert(angle_interval_coarse_align > 0 && angle_interval_coarse_align <= 25);
    assert(angle_interval_fine_align > 0 && angle_interval_fine_align <= 10);

    // Original source image
    int orig_src_width = orig_source_img_mat_.cols;
    int orig_src_height = orig_source_img_mat_.rows;
    int orig_tmp_width = orig_template_img_mat_.cols;
    int orig_tmp_height = orig_template_img_mat_.rows;
    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    // Down-sampling for source image
    cv::Mat pyr_src_img;
    cv::pyrDown(source_img, pyr_src_img, cv::Size(source_img.cols/2, source_img.rows/2));
    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    pyr_source_img_mat_ = pyr_src_img;
    int pyr_src_width = pyr_source_img_mat_.cols;
    int pyr_src_height = pyr_source_img_mat_.rows;
    int pyr_tmp_width = pyr_template_img_mat_.cols;
    int pyr_tmp_height = pyr_template_img_mat_.rows;
    if(dm::log){
        checkCUDAandSysInfo();
        hipDeviceSynchronize();
    }
    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    image_proc->SkipBlurContour(false, false);
    orig_source_contours_mat_.release();


    image_proc->SkipBlurContour(false, false);
    createSrcNonZeroMat(pyr_src_img, preprocessed_source_pyr, pyr_tmp_width, pyr_tmp_height,
                        pyr_src_width, pyr_src_height, pyr_path);
    if(expand_img_){
        pyr_src_width = pyr_source_img_mat_.cols + pyr_tmp_width;
        pyr_src_height = pyr_source_img_mat_.rows + pyr_tmp_height;
    }
    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    // Check if the nonzero parts are detected
    if(preprocessed_source_pyr.length_source_candidate < 1){
        t_execute = 0;
        return 0;
    }

    if(dm::log){
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_end);
        host_elapsed = clock_diff (&tp_start, &tp_end);
        clock_gettime(CLOCK_MONOTONIC, &tp_start);

        lw << "[DEBUG] Time elapsed - createSrcNonZeroMat: " << host_elapsed << "s" << lw.endl;
    }
    ////////////////////////////////// Prepare Pointer Variables //////////////////////////////////
    int* template_num_non_zero = NULL;

    // CUDA memory
    short* cuda_flat_template_non_zero_col = NULL;
    short* cuda_flat_template_non_zero_row = NULL;
    short* cuda_source_candidate_col = NULL;
    short* cuda_source_candidate_row = NULL;
    // Jang 20220330
    float* cuda_flat_template_weight_pyr = NULL;

    float* cuda_flat_score_map = NULL;
    int* cuda_flat_score_map_index = NULL;
    float* cuda_flat_source_dist_transfrom = NULL;
    int* cuda_template_num_non_zero = NULL;

    // Used in the sorting process but old version.
    float* cuda_score_candidates = NULL;
    int* cuda_score_candidates_index = NULL;
    float* cuda_score_candidates_result_2nd = NULL;
    int* cuda_score_candidates_result_index_2nd = NULL;
    float* cuda_score_candidates_result_3rd = NULL;
    int* cuda_score_candidates_result_index_3rd = NULL;

    float* score_candidates_result = NULL;
    int* score_candidates_result_index = NULL;
    short* orig_src_cand_col = NULL;
    short* orig_src_cand_row = NULL;

    ////////////////////////////////// Memory Trans. {Host to Device} //////////////////////////////////
    // Copy memories into CUDA
    hipMalloc(&cuda_source_candidate_col, preprocessed_source_pyr.length_source_candidate * sizeof(short));
    hipMemcpy(cuda_source_candidate_col, preprocessed_source_pyr.source_candidate_col,
               preprocessed_source_pyr.length_source_candidate * sizeof(short), hipMemcpyHostToDevice);
    hipMalloc(&cuda_source_candidate_row, preprocessed_source_pyr.length_source_candidate * sizeof(short));
    hipMemcpy(cuda_source_candidate_row, preprocessed_source_pyr.source_candidate_row,
               preprocessed_source_pyr.length_source_candidate * sizeof(short), hipMemcpyHostToDevice);

    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    // Score
    /// No need to make entire score map. Because the only non-zero positions will be computed.
    /// (Reducing the resource and process time.)
    int score_mem_size = trained_template_pyr.num_templates * preprocessed_source_pyr.length_source_candidate;

    hipMalloc(&cuda_flat_score_map, score_mem_size * sizeof(float));
    hipMalloc(&cuda_flat_score_map_index, score_mem_size * sizeof(int));

    // Set templates
    hipMalloc(&cuda_flat_template_non_zero_col, trained_template_pyr.total_memory_size * sizeof(short));
    hipMalloc(&cuda_flat_template_non_zero_row, trained_template_pyr.total_memory_size * sizeof(short));
    hipMemcpy(cuda_flat_template_non_zero_col, trained_template_pyr.flat_template_non_zero_col_,
               trained_template_pyr.total_memory_size * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(cuda_flat_template_non_zero_row, trained_template_pyr.flat_template_non_zero_row_,
               trained_template_pyr.total_memory_size * sizeof(short), hipMemcpyHostToDevice);

    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    // Jang 20220330
    hipMalloc(&cuda_flat_template_weight_pyr, trained_template_pyr.total_memory_size * sizeof(float));
    hipMemcpy(cuda_flat_template_weight_pyr, trained_template_pyr.flat_template_weight_,
               trained_template_pyr.total_memory_size * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&cuda_flat_source_dist_transfrom, pyr_src_width * pyr_src_height * sizeof(float));
    hipMemcpy(cuda_flat_source_dist_transfrom, preprocessed_source_pyr.flat_source_dist_transfrom,
               pyr_src_width * pyr_src_height * sizeof(float), hipMemcpyHostToDevice);

    template_num_non_zero = trained_template_pyr.num_non_zeros;
    hipMalloc(&cuda_template_num_non_zero, trained_template_pyr.num_templates * sizeof(int));
    hipMemcpy(cuda_template_num_non_zero, template_num_non_zero,
               trained_template_pyr.num_templates * sizeof(int), hipMemcpyHostToDevice);
    //////////////////////////////////////////////////////////////////////////////////////////////////////


    ////////////////////////////////// CUDA: Set the CUDA memory block //////////////////////////////////
    int block_size = 32;    // max: 32
    dim3 threads_in_a_block(block_size, block_size);    // 16x16 threads
    int num_blocks_w = (int)((preprocessed_source_pyr.source_num_candidate_width + block_size - 1) / block_size);
    int num_blocks_h = (int)((preprocessed_source_pyr.source_num_candidate_height + block_size - 1) / block_size);
    dim3 num_blocks(num_blocks_w, num_blocks_h);

    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    // short length_source_candidate, short source_num_candidate_width, short source_num_candidate_height,
    // these should be computed for threads....
    // get total length
    // separate them

    // Jang 20220330
    ChamferMatch<<<num_blocks, threads_in_a_block>>>(cuda_flat_source_dist_transfrom,
                                 cuda_source_candidate_col, cuda_source_candidate_row,
                                 pyr_src_width, pyr_src_height,
                                 trained_template_pyr.num_templates,
                                 cuda_flat_template_non_zero_col, cuda_flat_template_non_zero_row, cuda_template_num_non_zero,
                                 cuda_flat_template_weight_pyr, use_weight,
                                 preprocessed_source_pyr.length_source_candidate, preprocessed_source_pyr.source_num_candidate_width,
                                 preprocessed_source_pyr.source_num_candidate_height,
                                 cuda_flat_score_map, cuda_flat_score_map_index);

    if(dm::log){
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_end);
        host_elapsed = clock_diff (&tp_start, &tp_end);
        clock_gettime(CLOCK_MONOTONIC, &tp_start);

        lw << "[DEBUG] Time elapsed - ChamferMatch: " << host_elapsed << "s" << lw.endl;
    }


    // Do score sorting


    int divider;              // This is the number of threads.
    int num_score_blocks;

    int target_num_scores;     // each thread will collect top 10(target_num_scores) scores (which are lowest scores)
                                // // if a target_num_score is 10, one thread sort out 10 sorted values.
    int num_scores = score_mem_size;    // the number of templates * length_source_candidate

    int num_result_scores;

    lw << "[DEBUG] num_scores(num_templates x candidates): " << num_scores << lw.endl;
#ifdef DEBUG_MODE
    lw << "[DEBUG] trained_template_pyr.num_templates: " << trained_template_pyr.num_templates << lw.endl;
#endif

#ifdef DEBUG_MODE
    lw << "[DEBUG] preprocessed_source_pyr.length_source_candidate: " << preprocessed_source_pyr.length_source_candidate << lw.endl;
#endif

    bool is_fine_detection = false;
    bool old_sorting_method = false;
    int steps = 3;
    std::vector<int> target_num_score_list; // if a target_num_score is 10, one thread sort out 10 sorted values.
    std::vector<int> divider_list;
    target_num_score_list.resize(steps);
    divider_list.resize(steps);

    target_num_score_list.at(0) = 10;
    divider_list.at(0) = 1000;
    target_num_score_list.at(1) = 10;
    divider_list.at(1) = 100;
    target_num_score_list.at(2) = 10;
    divider_list.at(2) = 40;

    num_result_scores = target_num_score_list.at(target_num_score_list.size() - 1) * divider_list.at(divider_list.size() - 1);

    // Old one
    if(old_sorting_method){
        // 384 cores
        // 1000 threads / 384 = 2.xx
        // block size = 3
        target_num_scores = target_num_score_list.at(0);
        divider = divider_list.at(0);
        block_size = 32;
        num_result_scores = target_num_scores * divider;
        hipMalloc(&cuda_score_candidates, num_result_scores * sizeof(float));
        hipMalloc(&cuda_score_candidates_index, num_result_scores * sizeof(int));

        threads_in_a_block = dim3(block_size);
        num_score_blocks = (int)((divider + block_size - 1) / block_size);
        num_blocks = dim3(num_score_blocks);
        SortChamferScores<<<num_blocks, threads_in_a_block>>>(cuda_flat_score_map, cuda_flat_score_map_index,
                                     cuda_score_candidates, cuda_score_candidates_index,
                                     num_scores,
                                     target_num_scores, divider,
                                    threshold_score_py, max_score_py);

        // Second sorting
        num_scores  = num_result_scores;

        target_num_scores = target_num_score_list.at(1);
        divider = divider_list.at(1);
        num_result_scores = target_num_scores * divider;

        hipMalloc(&cuda_score_candidates_result_2nd, num_result_scores * sizeof(float));
        hipMalloc(&cuda_score_candidates_result_index_2nd, num_result_scores * sizeof(int));

        block_size = 2;
        threads_in_a_block = dim3(block_size);
        num_score_blocks = (int)((divider + block_size - 1) / block_size);
        num_blocks = dim3(num_score_blocks);
        SortChamferScores<<<num_blocks, threads_in_a_block>>>(cuda_score_candidates, cuda_score_candidates_index,
                                     cuda_score_candidates_result_2nd, cuda_score_candidates_result_index_2nd,
                                     num_scores,
                                     target_num_scores, divider,
                                     threshold_score_py, max_score_py);


        num_scores  = num_result_scores;

        target_num_scores = target_num_score_list.at(2);
        divider = divider_list.at(2);
        num_result_scores = target_num_scores * divider;

        hipMalloc(&cuda_score_candidates_result_3rd, num_result_scores * sizeof(float));
        hipMalloc(&cuda_score_candidates_result_index_3rd, num_result_scores * sizeof(int));

        block_size = 1;
        threads_in_a_block = dim3(block_size);
        num_score_blocks = (int)((divider + block_size - 1) / block_size);
        num_blocks = dim3(num_score_blocks);
        SortChamferScores<<<num_blocks, threads_in_a_block>>>(cuda_score_candidates_result_2nd, cuda_score_candidates_result_index_2nd,
                                     cuda_score_candidates_result_3rd, cuda_score_candidates_result_index_3rd,
                                     num_scores,
                                     target_num_scores, divider,
                                     threshold_score_py, max_score_py);


        score_candidates_result = new float[num_result_scores];
        score_candidates_result_index = new int[num_result_scores];
        hipMemcpy(score_candidates_result, cuda_score_candidates_result_3rd, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(score_candidates_result_index, cuda_score_candidates_result_index_3rd, num_result_scores * sizeof(int), hipMemcpyDeviceToHost);

        FreeCudaMemory(&cuda_score_candidates);
        FreeCudaMemory(&cuda_score_candidates_index);
        FreeCudaMemory(&cuda_score_candidates_result_2nd);
        FreeCudaMemory(&cuda_score_candidates_result_index_2nd);
        FreeCudaMemory(&cuda_score_candidates_result_3rd);
        FreeCudaMemory(&cuda_score_candidates_result_index_3rd);

        FreeCudaMemory(&cuda_flat_score_map);
        FreeCudaMemory(&cuda_flat_score_map_index);
    }
    else{
        hipMalloc(&cuda_score_candidates, num_result_scores * sizeof(float));
        hipMalloc(&cuda_score_candidates_index, num_result_scores * sizeof(int));

        SortScores(cuda_flat_score_map, cuda_flat_score_map_index,
                    cuda_score_candidates, cuda_score_candidates_index,
                    num_scores, steps,
                   target_num_score_list, divider_list,
                    is_fine_detection);

        score_candidates_result = new float[num_result_scores];
        score_candidates_result_index = new int[num_result_scores];
        hipMemcpy(score_candidates_result, cuda_score_candidates, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(score_candidates_result_index, cuda_score_candidates_index, num_result_scores * sizeof(int), hipMemcpyDeviceToHost);

        FreeCudaMemory(&cuda_score_candidates);
        FreeCudaMemory(&cuda_score_candidates_index);
    }

    if(dm::log){
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_end);
        host_elapsed = clock_diff (&tp_start, &tp_end);
        clock_gettime(CLOCK_MONOTONIC, &tp_start);
        lw << "[DEBUG] Time elapsed - Sorting: " << host_elapsed << "s" << lw.endl;
    //    for (unsigned int i = 0; i < num_result_scores; i++){
    //        int source_target = score_candidates_result_index[i] % preprocessed_source_pyr.length_source_candidate;
    //        int source_target_rot = score_candidates_result_index[i] / preprocessed_source_pyr.length_source_candidate;
    //        short x__= preprocessed_source_pyr.source_candidate_col[source_target];
    //        short y__ = preprocessed_source_pyr.source_candidate_row[source_target];
    //        lw << "[DEBUG] Score: " << score_candidates_result[i] << lw.endl;
    //        lw << "[DEBUG] source_target: " << source_target << lw.endl;
    //        lw << "[DEBUG] source_target_rot: " << source_target_rot << lw.endl;
    //        lw << "[DEBUG] x__: " << x__ << lw.endl;
    //        lw << "[DEBUG] y__: " << y__ << lw.endl;
    //    }
    }

    template_num_non_zero = NULL;
    // Free memory
    FreeCudaMemory(&cuda_flat_template_non_zero_col);
    FreeCudaMemory(&cuda_flat_template_non_zero_row);
    FreeCudaMemory(&cuda_flat_template_weight_pyr);
    FreeCudaMemory(&cuda_source_candidate_col);
    FreeCudaMemory(&cuda_source_candidate_row);

    FreeCudaMemory(&cuda_flat_source_dist_transfrom);
    FreeCudaMemory(&cuda_template_num_non_zero);



    // ============================================================================================
    // ================================= Fineturn on Position =====================================
    /// From top 10 to do Chamfer matching in original domain
    // Create the non-zero mat for original source image
    orig_source_contours_mat_.release();
    createSrcNonZeroMat(orig_source_img_mat_, preprocessed_source, orig_tmp_width, orig_tmp_height,
                        orig_src_width, orig_src_height, orig_path);
    if(expand_img_){
        orig_src_width = orig_source_img_mat_.cols + orig_tmp_width;
        orig_src_height = orig_source_img_mat_.rows + orig_tmp_height;
    }
    if(dm::log){
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_end);
        host_elapsed = clock_diff (&tp_start, &tp_end);
        clock_gettime(CLOCK_MONOTONIC, &tp_start);

        lw << "[DEBUG] Time elapsed - createSrcNonZeroMat: " << host_elapsed << "s" << lw.endl;
        lw << "[DEBUG] preprocessed_source_pyr.length_source_candidate: " << preprocessed_source_pyr.length_source_candidate << lw.endl;
        lw << "[DEBUG] num_result_scores: " << num_result_scores << lw.endl;
    }
    // Find the original poision of top 10 coordinates
    // target_num_scores = 10; Because top 10 from down-sampling domain
    int pyr2 = (int)pow(2, num_pyramid);
    unsigned int count = 0;
    int orig_tot_mem_size = num_result_scores * pyr2 * pyr2;
    int filtered_tot_mem_size = 0;

    // num_result_scores:100
    // * pyr2: 2 * pyr2: 2 *




    std::vector<cv::Point> filter_target;
    filter_target.resize(num_result_scores);
    bool do_pass = false;
    for (unsigned int i = 0; i < num_result_scores; i++){
        int source_target = score_candidates_result_index[i] % preprocessed_source_pyr.length_source_candidate;
        short fitx = preprocessed_source_pyr.source_candidate_col[source_target] * pyr2;
        short fity = preprocessed_source_pyr.source_candidate_row[source_target] * pyr2;
//        lw << "[DEBUG] i: " << (int)i << ", source_target: " << source_target << ", (" << preprocessed_source_pyr.source_candidate_col[source_target] << ", " << preprocessed_source_pyr.source_candidate_row[source_target] <<  ");  ";

        do_pass = false;
        for (int j = i - 1; j >= 0; j--){
            if(filter_target.at(j).x == fitx && filter_target.at(j).y == fity){
//                lw << "[DEBUG] dupl: " << (int)fitx << ", " << (int)fity << lw.endl;
                do_pass = true;
                break;
            }
        }
        if(!do_pass){
            filter_target.at(i).x = fitx;
            filter_target.at(i).y = fity;
            count++;
        }
        else{
            filter_target.at(i).x = 0;
            filter_target.at(i).y = 0;
        }

    }
    orig_tot_mem_size = count * pyr2 * pyr2;
    orig_src_cand_col = new short[orig_tot_mem_size];
    orig_src_cand_row = new short[orig_tot_mem_size];
#ifdef DEBUG_MODE
    lw << "[DEBUG] orig_tot_mem_size: " << orig_tot_mem_size << lw.endl;
#endif
    count = 0;
    for (unsigned int i = 0; i < num_result_scores; i++){
        short fitx = filter_target.at(i).x;
        short fity = filter_target.at(i).y;
        if(fitx == 0 && fity == 0) continue;

        for (unsigned int upx = 0; upx < pyr2; upx++){      // #FIXME: upx++ => upx+=pow(2, pyrNum-i)
            for (unsigned int upy = 0; upy < pyr2; upy++){

                orig_src_cand_col[count] = fitx + upx;
                orig_src_cand_row[count] = fity + upy;

//                lw << "[DEBUG]:(" << orig_src_cand_col[count] << ", " << orig_src_cand_row[count] << "); Score:" << score_candidates_result[i] << lw.endl;

                count++;
            }
        }
    }
#ifdef DEBUG_MODE
    lw << "[DEBUG] count: " << (int)count << lw.endl;
#endif
    filter_target.clear();

/*
    count = 0;
    for (unsigned int i = 0; i < num_result_scores; i++){
        int source_target = score_candidates_result_index[i] % preprocessed_source_pyr.length_source_candidate;
        short fitx = preprocessed_source_pyr.source_candidate_col[source_target] * pyr2;
        short fity = preprocessed_source_pyr.source_candidate_row[source_target] * pyr2;
        lw << "[DEBUG] i: " << (int)i << ", source_target: " << source_target << ", (" << preprocessed_source_pyr.source_candidate_col[source_target] << ", " << preprocessed_source_pyr.source_candidate_row[source_target] <<  ");  ";
        for (unsigned int upx = 0; upx < pyr2; upx++){      // #FIXME: upx++ => upx+=pow(2, pyrNum-i)
            for (unsigned int upy = 0; upy < pyr2; upy++){
                orig_src_cand_col[count] = fitx + upx;
                orig_src_cand_row[count] = fity + upy;
                lw << "[DEBUG]:(" << orig_src_cand_col[count] << ", " << orig_src_cand_row[count] << "); Score:" << score_candidates_result[i] << lw.endl;
                count++;
            }
        }
    }
 */


    FreeMemory(&preprocessed_source_pyr.flat_score_map);
//    FreeMemory(&preprocessed_source_pyr.flat_source_dist_transfrom);  // This is already released
    FreeMemory(&preprocessed_source_pyr.source_candidate_col);
    FreeMemory(&preprocessed_source_pyr.source_candidate_row);
    FreeMemory(&score_candidates_result);
    FreeMemory(&score_candidates_result_index);

    if(count == 0){
        lw << "[FAIL] No template is detected"
           << ", File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;

        FreeMemory(&orig_src_cand_col);
        FreeMemory(&orig_src_cand_row);
        FreeMemory(&preprocessed_source.flat_score_map);
        FreeMemory(&preprocessed_source.source_candidate_col);
        FreeMemory(&preprocessed_source.source_candidate_row);
        t_execute = 0;
        return 0;
    }

    // Declare the CUDA paras.
    int* template_num_non_zero_orig = NULL;

    // CUDA memory
    short* cuda_flat_template_non_zero_col_orig = NULL;
    short* cuda_flat_template_non_zero_row_orig = NULL;
    short* cuda_source_candidate_col_orig = NULL;
    short* cuda_source_candidate_row_orig = NULL;

    // Jang 20220330
    float* cuda_flat_template_weight_orig = NULL;

    float* cuda_flat_score_map_orig = NULL;
    int* cuda_flat_score_map_index_orig = NULL;
    float* cuda_flat_source_dist_transfrom_orig = NULL;
    int* cuda_template_num_non_zero_orig = NULL;

    float* cuda_score_candidates_orig = NULL;
    int* cuda_score_candidates_index_orig = NULL;

    float* score_candidates_result_orig = NULL;
    int* score_candidates_result_index_orig = NULL;

    ////////////////////////////////// Memory Trans. {Host to Device} //////////////////////////////////
    // Put to CUDA
    hipMalloc(&cuda_source_candidate_col_orig, orig_tot_mem_size * sizeof(short));
    hipMemcpy(cuda_source_candidate_col_orig, orig_src_cand_col,
               orig_tot_mem_size * sizeof(short), hipMemcpyHostToDevice);
    hipMalloc(&cuda_source_candidate_row_orig, orig_tot_mem_size * sizeof(short));
    hipMemcpy(cuda_source_candidate_row_orig, orig_src_cand_row,
               orig_tot_mem_size * sizeof(short), hipMemcpyHostToDevice);

    // Score
    int orig_score_mem_size = trained_template.num_templates * orig_tot_mem_size;

    hipMalloc(&cuda_flat_score_map_orig, orig_score_mem_size * sizeof(float));
    hipMalloc(&cuda_flat_score_map_index_orig, orig_score_mem_size * sizeof(int));

    // Set templates
    hipMalloc(&cuda_flat_template_non_zero_col_orig, trained_template.total_memory_size * sizeof(short));
    hipMalloc(&cuda_flat_template_non_zero_row_orig, trained_template.total_memory_size * sizeof(short));
    hipMemcpy(cuda_flat_template_non_zero_col_orig, trained_template.flat_template_non_zero_col_,
               trained_template.total_memory_size * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(cuda_flat_template_non_zero_row_orig, trained_template.flat_template_non_zero_row_,
               trained_template.total_memory_size * sizeof(short), hipMemcpyHostToDevice);
    hipMalloc(&cuda_flat_source_dist_transfrom_orig, orig_src_width * orig_src_height * sizeof(float));

    // Jang 20220330
    hipMalloc(&cuda_flat_template_weight_orig, trained_template.total_memory_size * sizeof(float));
    hipMemcpy(cuda_flat_template_weight_orig, trained_template.flat_template_weight_,
               trained_template.total_memory_size * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(cuda_flat_source_dist_transfrom_orig, preprocessed_source.flat_source_dist_transfrom,
               orig_src_width * orig_src_height * sizeof(float), hipMemcpyHostToDevice);

    template_num_non_zero_orig = trained_template.num_non_zeros;
    hipMalloc(&cuda_template_num_non_zero_orig, trained_template.num_templates * sizeof(int));
    hipMemcpy(cuda_template_num_non_zero_orig, template_num_non_zero_orig,
               trained_template.num_templates * sizeof(int), hipMemcpyHostToDevice);
    //////////////////////////////////////////////////////////////////////////////////////////////////////


    ////////////////////////////////// CUDA: Set the CUDA memory block //////////////////////////////////
    block_size = 32;    // max: 32
    threads_in_a_block = dim3(block_size, block_size);    // 16x16 threads
    num_blocks_w = (int)((preprocessed_source.source_num_candidate_width + block_size - 1) / block_size);
    num_blocks_h = (int)((preprocessed_source.source_num_candidate_height + block_size - 1) / block_size);
    num_blocks = dim3(num_blocks_w, num_blocks_h);

    int cand_width = (int)(sqrt(orig_tot_mem_size) + 0.5);
    int cand_height = (int)((float)orig_tot_mem_size / (float)cand_width + 0.5);
#ifdef DEBUG_MODE
    lw << "[DEBUG] cand_width: " << cand_width<< lw.endl;
    lw << "[DEBUG] cand_height: " << cand_height<< lw.endl;
    lw << "[DEBUG] (int)sqrt(orig_tot_mem_size)+1: " << (int)sqrt(orig_tot_mem_size)+1<< lw.endl;
#endif

    // Jang 20220330
//    use_weight_ = 1;
    // Do Chamfer Matching in original domain
    ChamferMatch<<<num_blocks, threads_in_a_block>>>(cuda_flat_source_dist_transfrom_orig,
                                                        cuda_source_candidate_col_orig, cuda_source_candidate_row_orig,
                                                        orig_src_width, orig_src_height,
                                                        trained_template.num_templates,
                                                        cuda_flat_template_non_zero_col_orig, cuda_flat_template_non_zero_row_orig, cuda_template_num_non_zero_orig,
                                                        cuda_flat_template_weight_orig, use_weight,
                                                        orig_tot_mem_size, cand_width, cand_height,
                                                        cuda_flat_score_map_orig, cuda_flat_score_map_index_orig);

#ifdef DEBUG_MODE
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &tp_end);
    host_elapsed = clock_diff (&tp_start, &tp_end);
    clock_gettime(CLOCK_MONOTONIC, &tp_start);
    lw << "[DEBUG] Time elapsed - ChamferMatch: " << host_elapsed << "s" << lw.endl;
#endif
    FreeMemory(&preprocessed_source.flat_score_map);
//    FreeMemory(&preprocessed_source.flat_source_dist_transfrom);  // This is already released
    FreeMemory(&preprocessed_source.source_candidate_col);
    FreeMemory(&preprocessed_source.source_candidate_row);


    // Do Chamfer sorting
    // Fineturn position sorting
    num_scores = orig_score_mem_size;

    lw << "[DEBUG] num_scores[fine](num_templates x candidates): " << num_scores << lw.endl;

#ifdef DEBUG_MODE
//    float* TEST_score_candidates_result = NULL;
//    int* TEST_score_candidates_result_index = NULL;
//    // ex) orig_tot_mem_size = 40
//    TEST_score_candidates_result = new float[num_scores];
//    TEST_score_candidates_result_index = new int[num_scores];
//    hipMemcpy(TEST_score_candidates_result, cuda_flat_score_map_orig, num_scores * sizeof(float), hipMemcpyDeviceToHost);(num_templates x candidates)
//    hipMemcpy(TEST_score_candidates_result_index, cuda_flat_score_map_index_orig, num_scores * sizeof(int), hipMemcpyDeviceToHost);

//    for(int i = 0; i < num_scores; i ++){
//        if(TEST_score_candidates_result[i] < 0.1){
//            lw << "[DEBUG] Score: " << TEST_score_candidates_result[i] << ", " << TEST_score_candidates_result_index[i] << lw.endl;
//        }

//    }

//    FreeMemory(&TEST_score_candidates_result);
//    TEST_score_candidates_result = NULL;
//    FreeMemory(&TEST_score_candidates_result_index);
//    TEST_score_candidates_result_index = NULL;
#endif

    //
    is_fine_detection = true;
    old_sorting_method = false;
    steps = 3;

    target_num_score_list.resize(steps);
    divider_list.resize(steps);

    target_num_score_list.at(0) = 10;
    divider_list.at(0) = 1000;
    target_num_score_list.at(1) = 10;
    divider_list.at(1) = 100;
    target_num_score_list.at(2) = 30;
    divider_list.at(2) = 1;

    num_result_scores = target_num_score_list.at(target_num_score_list.size() - 1) * divider_list.at(divider_list.size() - 1);


    //
    if(old_sorting_method){
        target_num_scores = 10;
        divider =1;
        num_result_scores = target_num_scores * divider;

        hipMalloc(&cuda_score_candidates_orig, num_result_scores * sizeof(float));
        hipMalloc(&cuda_score_candidates_index_orig, num_result_scores * sizeof(int));

        block_size = 1;
        threads_in_a_block = dim3(block_size);
        num_score_blocks = (int)((divider + block_size - 1) / block_size);
        num_blocks = dim3(num_score_blocks);


        SortChamferScores<<<num_blocks, threads_in_a_block>>>(cuda_flat_score_map_orig, cuda_flat_score_map_index_orig,
                                     cuda_score_candidates_orig, cuda_score_candidates_index_orig,
                                     num_scores,
                                     target_num_scores, divider,
                                     threshold_score_orig, max_score_orig);
        score_candidates_result_orig = new float[num_result_scores];
        score_candidates_result_index_orig = new int[num_result_scores];
        hipMemcpy(score_candidates_result_orig, cuda_score_candidates_orig, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(score_candidates_result_index_orig, cuda_score_candidates_index_orig, num_result_scores * sizeof(int), hipMemcpyDeviceToHost);

        FreeCudaMemory(&cuda_flat_score_map_orig);
        FreeCudaMemory(&cuda_flat_score_map_index_orig);
    }
    else{
        hipMalloc(&cuda_score_candidates_orig, num_result_scores * sizeof(float));
        hipMalloc(&cuda_score_candidates_index_orig, num_result_scores * sizeof(int));

        SortScores(cuda_flat_score_map_orig, cuda_flat_score_map_index_orig,
                    cuda_score_candidates_orig, cuda_score_candidates_index_orig,
                    num_scores, steps,
                   target_num_score_list, divider_list,
                    is_fine_detection);

        score_candidates_result_orig = new float[num_result_scores];
        score_candidates_result_index_orig = new int[num_result_scores];
        hipMemcpy(score_candidates_result_orig, cuda_score_candidates_orig, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(score_candidates_result_index_orig, cuda_score_candidates_index_orig, num_result_scores * sizeof(int), hipMemcpyDeviceToHost);
    }


#ifdef DEBUG_MODE
    for(int i = 0; i < num_result_scores; i ++){
        lw << "[DEBUG] Score(fine): " << score_candidates_result_orig[i] << lw.endl;
    }
#endif

    bool cuda_free_mem_result;
    cuda_free_mem_result = FreeCudaMemory(&cuda_score_candidates_orig);

#ifdef DEBUG_MODE
    // To check whether the memory addresses are same or not.
    // If cuda memory is copied to the Host twice to difference host memory space, the addresses are different.
    if(!cuda_free_mem_result){
        float* TEST_cuda_score_candidates_orig_3 = NULL;
        TEST_cuda_score_candidates_orig_3 = new float[num_result_scores];
        hipMemcpy(TEST_cuda_score_candidates_orig_3, cuda_score_candidates_orig, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);

        for(int i = 0; i < num_result_scores; i ++){
            lw << "[DEBUG] cuda_score_candidates_orig: " << TEST_cuda_score_candidates_orig_3[i] << lw.endl;
        }
        std::cout << "[DEBUG] mem addr: " << TEST_cuda_score_candidates_orig_3 << std::endl ;

        float* TEST_cuda_score_candidates_orig_2 = NULL;
        TEST_cuda_score_candidates_orig_2 = new float[num_result_scores];
        hipMemcpy(TEST_cuda_score_candidates_orig_2, cuda_score_candidates_orig, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);

        for(int i = 0; i < num_result_scores; i ++){
            lw << "[DEBUG] cuda_score_candidates_orig: " << TEST_cuda_score_candidates_orig_2[i] << lw.endl;
        }
        std::cout << "[DEBUG] mem addr: " << TEST_cuda_score_candidates_orig_2 << std::endl ;

        FreeMemory(&TEST_cuda_score_candidates_orig_3);
        FreeMemory(&TEST_cuda_score_candidates_orig_2);
    }
    // BUT
    // If cuda memory is copied to the Host and "Free" the memory.
    // And then copy it again to another host memory space, the addresses are same.
    if(!cuda_free_mem_result){
        float* TEST_cuda_score_candidates_orig = NULL;
        TEST_cuda_score_candidates_orig = new float[num_result_scores];
        hipMemcpy(TEST_cuda_score_candidates_orig, cuda_score_candidates_orig, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);

        for(int i = 0; i < num_result_scores; i ++){
            lw << "[DEBUG] cuda_score_candidates_orig: " << TEST_cuda_score_candidates_orig[i] << lw.endl;
        }
        std::cout << "[DEBUG] mem addr: " << TEST_cuda_score_candidates_orig << std::endl ;
        FreeMemory(&TEST_cuda_score_candidates_orig);


        float* TEST_cuda_score_candidates_orig_2 = NULL;
        TEST_cuda_score_candidates_orig_2 = new float[num_result_scores];
        hipMemcpy(TEST_cuda_score_candidates_orig_2, cuda_score_candidates_orig, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);

        for(int i = 0; i < num_result_scores; i ++){
            lw << "[DEBUG] cuda_score_candidates_orig: " << TEST_cuda_score_candidates_orig_2[i] << lw.endl;
        }
        std::cout << "[DEBUG] mem addr: " << TEST_cuda_score_candidates_orig_2 << std::endl ;
        FreeMemory(&TEST_cuda_score_candidates_orig_2);
    }
#endif
    cuda_free_mem_result = FreeCudaMemory(&cuda_score_candidates_index_orig);

#ifdef DEBUG_MODE
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &tp_end);
    host_elapsed = clock_diff (&tp_start, &tp_end);
    clock_gettime(CLOCK_MONOTONIC, &tp_start);
    lw << "[DEBUG] Time elapsed - Sorting: " << host_elapsed << "s" << lw.endl;
#endif
    // For DEBUG
//    for (unsigned int i = 0; i<num_result_scores; i++){
//        lw << ">> [DEBUG]:(" << score_candidates_result_index_orig[i] <<
//              "; Score:" << score_candidates_result_orig[i] << lw.endl;
//    }

    // Free memory
    template_num_non_zero_orig = NULL;

    FreeCudaMemory(&cuda_flat_template_non_zero_col_orig);
    FreeCudaMemory(&cuda_flat_template_non_zero_row_orig);
    FreeCudaMemory(&cuda_source_candidate_col_orig);
    FreeCudaMemory(&cuda_source_candidate_row_orig);
    // Jang 20220330
    FreeCudaMemory(&cuda_flat_template_weight_orig);

    FreeCudaMemory(&cuda_flat_source_dist_transfrom_orig);
    FreeCudaMemory(&cuda_template_num_non_zero_orig);


    // $$ END of Fineturn on Position


#ifdef DEBUG_MODE
    // ex) orig_tot_mem_size = 40
    for(int i = 0; i < num_result_scores; i ++){
        int template_rot = score_candidates_result_index_orig[i] / orig_tot_mem_size;
        int source_target = score_candidates_result_index_orig[i] % orig_tot_mem_size;

        // Printout results
        int target_x = orig_src_cand_col[source_target] + orig_tmp_width / 2;
        int target_y = orig_src_cand_row[source_target] + orig_tmp_height / 2;


        // (45 + -45)
        // (5 + (-20 / 1)) * 1
        // 20
        float detected_angle = ((float)template_rot * angle_interval_fine_align) + angle_min;

        lw << "[DEBUG] Score: " << score_candidates_result_orig[i] << lw.endl;
        lw << "The result: " << "[ " << target_x << ", "<< target_y << " ]";
        lw << "| ang: " << detected_angle<< " deg." << lw.endl;
    }

#endif

//    // Save the result
//    cv::Rect crop_roi = cv::Rect(orig_src_cand_col[source_target],
//                                 orig_src_cand_row[source_target],
//                                 orig_tmp_width, orig_tmp_height);
//    cv::Mat mask_mat = trained_template.templates.at(template_rot);
//    cv::cvtColor(mask_mat, mask_mat, cv::COLOR_GRAY2RGB);
//    cv::Mat mask;
//    cv::inRange(mask_mat, cv::Scalar(250, 250, 250), cv::Scalar(255, 255, 255), mask);
//    mask_mat.setTo(cv::Scalar(0, 0, 255), mask);
//    mask_mat.copyTo(source_img(crop_roi), mask_mat);

//    cv::line(source_img, cv::Point(target_x-10, target_y), cv::Point(target_x+10, target_y), cv::Scalar(0, 255, 0), 5, cv::LINE_AA);
//    cv::line(source_img, cv::Point(target_x, target_y-10), cv::Point(target_x, target_y+10), cv::Scalar(0, 255, 0), 5, cv::LINE_AA);

//    cv::imwrite("./sss/copied_source_img.png", source_img);
//    cv::imwrite("./sss/copied_mask_mat.png", mask_mat);



    int best_case = 0;
    double best_ncc = 0;
    cv::Mat copied_source_img = source_img.clone();
    int length_source_candidate = orig_tot_mem_size;
    int template_width = orig_tmp_width;
    int template_height = orig_tmp_height;
    short* source_candidate_col = orig_src_cand_col;
    short* source_candidate_row = orig_src_cand_row;
    cv::Mat source_canny_img;
    cv::Mat source_gray_img = copied_source_img.clone();
    cv::Mat rot_template_mat;
    std::vector<float> ncc_results_vec;

    try{
        // NCC computation preparation
        if(copied_source_img.channels() > 1)
            cv::cvtColor(copied_source_img, source_gray_img, cv::COLOR_BGR2GRAY);
    //    if(orig_source_contours_mat_.empty() || (orig_source_contours_mat_.rows != copied_source_img.rows)){
    //        if(!use_image_proc){
    //            // Reduce the noise with kernel a 3x3 before the canny
    //            cv::blur(source_gray_img, source_gray_img, cv::Size(3, 3));
    //            //Canny edge
    //            cv::Canny(source_gray_img, source_canny_img, 100, 100, 3);
    //        }
    //        else{
    //            source_canny_img = this->image_proc->GetImageBySavedInfo(source_gray_img, use_cuda_for_improc);
    //        }
    //    }
    //    else{
    //        source_canny_img = orig_source_contours_mat_.clone();
    //    }


        if(expand_img_){
            std::vector<int> expanded_xy_index;
            bool pass_compute = false;
            for(int j = 0; j < num_result_scores; j++){
                int source_target = score_candidates_result_index_orig[j] % length_source_candidate;
                pass_compute = false;
                for(int k = 0; k < expanded_xy_index.size(); k ++){
                    if(source_target == expanded_xy_index.at(k)){
                        pass_compute = true;
                    }
                }
                if(pass_compute){
                    continue;
                }

                expanded_xy_index.push_back(source_target);

                if(dm::log){
                    std::cout << "=================orig===========================" << std::endl;
                    std::cout << "source_target: " << source_target << std::endl;
                    std::cout << "score_candidates_result_index_orig[j]: " << score_candidates_result_index_orig[j] << std::endl;
                    std::cout << "source_candidate_col[source_target]: " << source_candidate_col[source_target] << std::endl;
                    std::cout << "source_candidate_row[source_target]: " << source_candidate_row[source_target] << std::endl;
                    std::cout << "============================================" << std::endl;
                }
                source_candidate_col[source_target] -= (short)(template_width / 2);
                source_candidate_row[source_target] -= (short)(template_height / 2);
                if(dm::log){
                    std::cout << "source_candidate_col[source_target]: " << source_candidate_col[source_target] << std::endl;
                    std::cout << "source_candidate_row[source_target]: " << source_candidate_row[source_target] << std::endl;
                }
            }
        }

        // Get the best case
        fs::create_directories(data_path + "results/");
        for(int j = 0; j < num_result_scores; j++){
            int template_scale_rot = score_candidates_result_index_orig[j] / length_source_candidate;
            int template_scale = template_scale_rot / num_angles;
            int template_rot = template_scale_rot % num_angles; // + angle_min; // Jang 20211203    // This is the position of rotation from the beginning. (ex) from angle_min.
            int source_target = score_candidates_result_index_orig[j] % length_source_candidate;

//            lw << "Index: " << score_candidates_result_index_orig[j] << ", Score: " << score_candidates_result_orig[j] << lw.endl;
//            lw << "[ " << source_candidate_col[source_target] << ", " << source_candidate_row[source_target] << " ]" << lw.endl;
//            lw << "Scale: " << template_scale << ", Rot: " << template_rot << lw.endl;

            // Mask
            cv::Rect crop_roi;
            cv::Rect crop_temp_roi;

            int crop_x;
            int crop_y;
            int crop_w;
            int crop_h;
            int crop_tmp_x;
            int crop_tmp_y;
            crop_x = source_candidate_col[source_target];
            crop_y = source_candidate_row[source_target];
            crop_w = template_width;
            crop_h = template_height;
            crop_tmp_x = 0;
            crop_tmp_y = 0;
            if(source_candidate_col[source_target] < 0){
                crop_x = 0;
                crop_w = template_width + source_candidate_col[source_target];
                crop_tmp_x = -source_candidate_col[source_target];
            }
            if(source_candidate_row[source_target] < 0){
                crop_y = 0;
                crop_h = template_height + source_candidate_row[source_target];
                crop_tmp_y = -source_candidate_row[source_target];
            }
            if((source_candidate_col[source_target] + template_width) >  orig_source_img_mat_.cols){
                crop_w = (orig_source_img_mat_.cols - source_candidate_col[source_target]);
            }
            if((source_candidate_row[source_target] + template_height) >  orig_source_img_mat_.rows){
                crop_h = (orig_source_img_mat_.rows - source_candidate_row[source_target]);
            }
            crop_roi = cv::Rect(crop_x, crop_y, crop_w, crop_h);
            crop_temp_roi = cv::Rect(crop_tmp_x, crop_tmp_y, crop_w, crop_h);

            if(dm::log){
                std::cout << "crop_roi: " << crop_roi << std::endl;
                std::cout << "crop_temp_roi: " << crop_temp_roi << std::endl;

                lw << "File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
                lw << "template_scale_rot: " << template_scale_rot << lw.endl;
                lw << "score_candidates_result_index_orig[j]: " << score_candidates_result_index_orig[j] << lw.endl;

            }
            cv::Size temp_resized_size;
            temp_resized_size.width = crop_w;
            temp_resized_size.height = crop_h;
            if(trained_template.templates_gray.size() > template_scale_rot){
                // NCC computation
//                rot_template_mat = trained_template.templates_gray.at(template_scale_rot)(crop_temp_roi);
                rot_template_mat = rot_template_mat.zeros(temp_resized_size, trained_template.templates_gray.at(template_scale_rot).type());
                trained_template.templates_gray.at(template_scale_rot)(crop_temp_roi).copyTo(rot_template_mat);
                // Average of source
                double avg_source;
                avg_source = cv::mean(source_gray_img(crop_roi))[0];

                // Average of template
                double avg_template;
                avg_template = cv::mean(rot_template_mat)[0];
                if(dm::match_img){
                    cv::imwrite(data_path + "results/" + std::to_string(j) + "_template_contour.png", trained_template.templates.at(template_scale_rot));
                    cv::imwrite(data_path + "results/" + std::to_string(j) + "_cropped_source_gray_img.png", source_gray_img(crop_roi));
                    cv::imwrite(data_path + "results/" + std::to_string(j) + "_rot_template_mat.png", rot_template_mat);
                }
                // Sum of child values
                double sum_numerator = 0;
                double sum_denominator_0 = 0;
                double sum_denominator_1 = 0;
                for(int r = 0; r < rot_template_mat.rows; r ++){
                    for(int c = 0; c < rot_template_mat.cols; c++){
                        double compute_val_0 = rot_template_mat.at<uchar>(r, c) - avg_template;
                        double compute_val_1 = source_gray_img.at<uchar>(r + crop_roi.y, c + crop_roi.x) - avg_source;
                        sum_numerator += (compute_val_0 * compute_val_1);

                        sum_denominator_0 += std::pow(compute_val_0, 2);
                        sum_denominator_1 += std::pow(compute_val_1, 2);

                    }
                }

                sum_denominator_0 = std::sqrt(sum_denominator_0);
                sum_denominator_1 = std::sqrt(sum_denominator_1);



                double ncc_result = 0;
                ncc_result = sum_numerator / (sum_denominator_0 * sum_denominator_1);
//                ncc_result = std::fabs(ncc_result);

    #ifdef DEBUG_MODE
                lw << "avg_source: " << avg_source << lw.endl;
                lw << "avg_template: " <<  avg_template << lw.endl;
                lw << "sum_denominator_0: " << sum_denominator_0 << lw.endl;
                lw << "sum_denominator_1: " << sum_denominator_1 << lw.endl;
                lw << "Score(ncc_result): " << ncc_result << lw.endl;
    #endif

                ncc_results_vec.push_back(ncc_result);
                if(best_ncc < ncc_result){
                    best_ncc = ncc_result;
                    best_case = j;
                }

//                // NCC using OpenCV
//                cv::Mat ncc_result_mat = cv::Mat::zeros(1, 1, CV_32FC1);
//                cv::matchTemplate(source_gray_img(crop_roi), rot_template_mat, ncc_result_mat, cv::TM_SQDIFF_NORMED);
//                lw << "Score(opencv TM_SQDIFF_NORMED): " << ncc_result_mat << lw.endl;
//                cv::matchTemplate(source_gray_img(crop_roi), rot_template_mat, ncc_result_mat, cv::TM_CCORR_NORMED);
//                lw << "Score(opencv TM_CCORR_NORMED): " << ncc_result_mat << lw.endl;
//                cv::matchTemplate(source_gray_img(crop_roi), rot_template_mat, ncc_result_mat, cv::TM_CCOEFF_NORMED);
//                lw << "Score(opencv TM_CCOEFF_NORMED): " << ncc_result_mat << lw.endl;
            }
        }
        lw << "best_case(ncc_result): " << best_case << lw.endl;

    #ifdef DEBUG_MODE
        lw << "num_angles: " << num_angles << ", angle_min: " << angle_min << lw.endl;
    #endif
        {
            int j = best_case;
            int template_scale_rot = score_candidates_result_index_orig[j] / length_source_candidate;
            int template_scale = template_scale_rot / num_angles;
            int template_rot = template_scale_rot % num_angles; // + angle_min; // Jang 20211203    // This is the position of rotation from the beginning. (ex) from angle_min.
            int source_target = score_candidates_result_index_orig[j] % length_source_candidate;

            float detected_angle = ((float)template_rot * angle_interval_fine_align) + angle_min;

            float score_percentage;
            score_percentage = (4.0 - pow(score_candidates_result_orig[j], 2)) / 4.0;
            if(score_percentage < 0) score_percentage = 0;

            lw << "------ Result ------" << lw.endl;
            lw << "Index: " << score_candidates_result_index_orig[j] << ", Score: " << score_candidates_result_orig[j] << ", NCC: " << best_ncc << lw.endl;
            lw << "[ " << source_candidate_col[source_target] << ", " << source_candidate_row[source_target] << " ]" << lw.endl;
            lw << "Scale: " << template_scale << ", Rot: " << template_rot << lw.endl;

            if(dm::match_img){
                // Mask
                cv::Rect crop_roi = cv::Rect(source_candidate_col[source_target] , source_candidate_row[source_target], template_width, template_height);

                cv::Mat mask_mat = trained_template.templates.at(template_scale_rot);
                lw << "--- template_scale_rot: " << template_scale_rot << lw.endl;
                lw << "mask_mat: " << mask_mat.cols << ", " << mask_mat.rows << lw.endl;
                cv::Mat color_mask = cv::Mat::zeros(mask_mat.size(), CV_8UC3);
                std::vector<cv::Point> non_zero_;
                cv::findNonZero(mask_mat, non_zero_);

                // Change pixels' color
                for(int p = 0; p < non_zero_.size(); p++){
                    cv::Vec3b& vec3b = color_mask.at<cv::Vec3b>(non_zero_.at(p).y, non_zero_.at(p).x) ;
                    vec3b[0] = 0;
                    vec3b[1] = 0;
                    vec3b[2] = 255;
                }

                lw << "File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
                lw << "color_mask: " << color_mask.cols << ", " << color_mask.rows << lw.endl;
                lw << "copied_source_img: " << copied_source_img.cols << ", " << copied_source_img.rows << lw.endl;
                lw << "mask_mat: " << mask_mat.cols << ", " << mask_mat.rows << lw.endl;
                lw << "crop_roi: " << crop_roi.width << ", " << crop_roi.height << lw.endl;

                try{
                    cv::cvtColor(copied_source_img, copied_source_img, cv::COLOR_GRAY2BGR);
                }
                catch(cv::Exception e){
                    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                    std::string error_what = e.what();
                    lw << error_what << lw.endl;
                }catch(...){
                    lw << "Error. " << "File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                }
                color_mask.copyTo(copied_source_img(crop_roi), mask_mat.clone());

                cv::imwrite(data_path + "results/" + std::to_string(j) + "_copied_source_img.png", copied_source_img);
                cv::imwrite(data_path + "results/" + std::to_string(j) + "_copied_mask_mat.png", color_mask);
            }

            MatchData match_data( 0, 0, 1.0f, .0f, .0f );
            match_data.x = source_candidate_col[source_target] + template_width / 2;   // center position
            match_data.y = source_candidate_row[source_target] + template_height / 2;  // center position
            match_data.angle = detected_angle; // unit: degree
            match_data.coef = best_ncc; // score_candidates_result_orig[j];
            match_data.scale = template_scale;

            // If the matched position is too close to the image boundaries, the matching failed.
            // if(match_data.x < (template_width / 4) || match_data.y < (template_height / 4) ||
//                    orig_source_img_mat_.cols - match_data.x < (template_width / 4) ||
//                    orig_source_img_mat_.rows - match_data.y < (template_height / 4) ){
//                 throw(__LINE__);
//             }
            // Jang 20221024
            // Disabled the function.
            if(match_data.x < (template_width / 2) || match_data.y < (template_height / 2) ||
                   orig_source_img_mat_.cols - match_data.x < (template_width / 2) ||
                   orig_source_img_mat_.rows - match_data.y < (template_height / 2) ){
                throw(__LINE__);
            }

            rslt_match.push_back(match_data);

            // Save the best candidates into a file. (the same file which is used in the RST matching)
            FILE *fp_result_p=fopen((data_path + result_matched_area_path).c_str(), "wb");
            {
                float angle_ = (detected_angle / 180 * PI) + (PI / 2);
                float radius_ = template_height / 2;
                // x1, y1: center position
                // x2, y2: template's boundary position to show angle
                int x1, x2, y1, y2;
                x1 = source_candidate_col[source_target] + template_width / 2;
                y1 = source_candidate_row[source_target] + template_height / 2;
                x2 = x1 + (std::cos(angle_)*radius_)+0.5;
                y2 = y1 - (sin(angle_)*radius_)+0.5;

                cv::Point pp0{x1, y1};
                cv::Point pp1{x2, y2};
                cv::Point p1{
                        x1+(cos(angle_)*((template_height-1)/2))+(cos(angle_+ (PI / 2))*((template_width-1)/2)),
                        y1-(sin(angle_)*((template_height-1)/2))-(sin(angle_+ (PI / 2))*((template_width-1)/2))
                };
                cv::Point p2{
                        (p1.x+(cos(angle_- (PI / 2))*template_width)),
                        (p1.y-(sin(angle_- (PI / 2))*template_width))
                };
                cv::Point p3{
                        (p2.x+(cos(angle_- PI)*template_height)),
                        p2.y-(sin(angle_- PI)*template_height)
                };
                cv::Point p4{
                        p1.x+(cos(angle_+ PI)*template_height),
                        p1.y-(sin(angle_+ PI)*template_height)
                };

//                fprintf(fp_result_p, "%u\n%u\n%u\n%u\n",pp0.x,pp0.y,pp1.x,pp1.y);
//                fprintf(fp_result_p, "%u\n%u\n%u\n%u\n",p1.x,p1.y,p2.x,p2.y);
//                fprintf(fp_result_p, "%u\n%u\n%u\n%u\n",p3.x,p3.y,p4.x,p4.y);

                fprintf(fp_result_p, "%d\n%d\n%d\n%d\n",pp0.x,pp0.y,pp1.x,pp1.y);
                fprintf(fp_result_p, "%d\n%d\n%d\n%d\n",p1.x,p1.y,p2.x,p2.y);
                fprintf(fp_result_p, "%d\n%d\n%d\n%d\n",p3.x,p3.y,p4.x,p4.y);
            }
            fclose(fp_result_p);    fp_result_p = NULL;



        }
    }
    catch(cv::Exception e){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;
        std::string error_what = e.what();
        lw << error_what << lw.endl;
        rslt_match.resize(1);
        rslt_match.at(0).coef = 0;
    }
    catch(int line){
        rslt_match.resize(1);
        rslt_match.at(0).coef = 0;
    }
    catch(...){
        lw << "Error. " << "File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;
        rslt_match.resize(1);
        rslt_match.at(0).coef = 0;
    }


#ifdef DEBUG_MODE
    hipDeviceSynchronize();
#endif
    clock_gettime(CLOCK_MONOTONIC, &tp_end);
    host_elapsed = clock_diff (&start_all, &tp_end);
    t_execute = host_elapsed;

    lw << "ChamferInference Done: " << host_elapsed << "s" << lw.endl;


    for(int j = 0; j < num_result_scores; j++){
        int template_scale_rot = score_candidates_result_index_orig[j] / length_source_candidate;
        int template_scale = template_scale_rot / num_angles;
        int template_rot = template_scale_rot % num_angles;
        int source_target = score_candidates_result_index_orig[j] % length_source_candidate;

        lw << "Index: " << score_candidates_result_index_orig[j] << ", Score: " << score_candidates_result_orig[j] << lw.endl;
        lw << "[ " << source_candidate_col[source_target] << ", " << source_candidate_row[source_target] << " ]" << lw.endl;
        lw << "Scale: " << template_scale << ", Rot: " << template_rot << lw.endl;

        if(ncc_results_vec.size() > j){
            lw << "Score(NCC): " << ncc_results_vec.at(j)<< lw.endl;
        }
    }

    // num of non zero
    short template_size = (template_width + template_width) / 2;
    float best_score = score_candidates_result_orig[best_case];
    lw << "template_size : " << template_size << lw.endl;
    lw << "best_score : " << best_score << lw.endl;
    lw << "best_ncc: " << best_ncc << lw.endl;  // rslt_match.at(0).coef
    if(0.5 > chamfer_score_threshold_ || chamfer_score_threshold_ > 10){
        chamfer_score_threshold_ = 3;
    }
    // if weight discount is small -> ncc threshold can be small
    if( 0.3 < best_ncc && best_ncc < 0.8
       && best_score < chamfer_score_threshold_){
        lw << "[WARNING] Chamfer matching score is high. But NCC score is low." << lw.endl;
    }
    else if(best_ncc < 0.8){
        fail_msg = "[FAILED]";
        fail_msg += "Failure 1: NCC matching score is lower than 0.8. ";

        if(score_candidates_result_orig[0] >= max_score_orig){
            fail_msg += "Failure 2: Chamfer matching score is too low. The current image may not have the shapes of the templates. Image processing parameters between templates and current images may be different. ";
        }

        lw << "[FAILED] Detection failed. Score: " << rslt_match.at(0).coef << lw.endl;
        lw << fail_msg << lw.endl;
        host_elapsed = 0;
        t_execute = host_elapsed;
        rslt_match.clear();
    }

    // Save result if matching is succeeded.
    if(host_elapsed > 0 && rslt_match.size() > 0){
        if(rslt_match.at(0).coef > 0.4){
            FILE *result_score=fopen((data_path + result_score_path).c_str(),"wb");
            for(int i = 0; i < 1; i++){
                MatchData& match_data = rslt_match.at(i);
                // Jang 20221020
                // Fixed wrong y-value problem.
                fprintf(result_score, "%2d:        (%4d,%4d);%12.1f%;%12.1f;%12.1f\n",
                        i+1, match_data.x, orig_src_height - match_data.y
                        , match_data.coef * 100.0
                        , _minScale + _scaleInterval * (float)match_data.scale, match_data.angle);
            }
            fclose(result_score); result_score = NULL;
        }
        else{
            // Jang 20221024
            host_elapsed = 0;
            t_execute = host_elapsed;
            rslt_match.clear();
            lw << "[FAILED] Detection Failed. " << "File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
        }
    }

    FreeMemory(&score_candidates_result_orig);
    FreeMemory(&score_candidates_result_index_orig);
    FreeMemory(&orig_src_cand_col);
    FreeMemory(&orig_src_cand_row);

    if(dm::log) {
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_end);
        host_elapsed = clock_diff (&tp_start, &tp_end);
        lw << "[DEBUG] Time elapsed - Saving results: " << host_elapsed << "s" << lw.endl;
    }
    return (float)(host_elapsed);
}

float CudaChamfer::ChamferInference2(cv::Mat source_img, bool use_weight){
#ifdef ENABLE_CHAMFER_VERSION2
//    // Jang 20220419
//    // Warming up function
//    int warmup_block_size =32;
//    dim3 warmup_threads_in_a_block(warmup_block_size, warmup_block_size);
//    int warmup_num_blocks_w = (int)((1000 + warmup_block_size - 1) / warmup_block_size);
//    int warmup_num_blocks_h = (int)((1000 + warmup_block_size - 1) / warmup_block_size);
//    dim3 warmup_num_blocks(warmup_num_blocks_w, warmup_num_blocks_h);
//    WarmUpGpu<<<warmup_num_blocks, warmup_threads_in_a_block>>>();

    // With Warming up function
    // Beginnig: 650 -> After 1 min: 360 // Normal: 240
    // Without Warming up function
    // Beginnig: 870 -> After 1 min: 355 // Normal: 230

//    use_weight_ = 1;
    // Jang 20220713
    const std::string pyr_path = data_path + "/pyr_source/";
    const std::string orig_path = data_path + "/orig_source/";
    const std::string result_path = data_path + "/results/";
    if(fs::exists(orig_path)){
       fs::remove_all(orig_path);
    }
    if(fs::exists(pyr_path)){
       fs::remove_all(pyr_path);
    }
    if(fs::exists(result_path)){
       fs::remove_all(result_path);
    }
    fs::create_directories(pyr_path);
    fs::create_directories(orig_path);
    fs::create_directories(result_path);

    // Remove result data
    if(fs::exists(data_path + result_score_path))
        fs::remove(data_path + result_score_path);

    // Jang 20220711
    // Should be modified
    lw << "[DEBUG] chamfer_method_: " << chamfer_method_
       << ", File " <<  __FILE__ <<  ", line " << __LINE__
       << ", in " << __func__ << lw.endl;

    fail_msg = "";
    if(trained_template_vec_.size() == 0){
        lw << "[FAIL] No template is detected"
           << ", File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;
        t_execute = 0;
        return 0;
    }
    for (int i = 0; i < trained_template_vec_.size(); ++i) {
        if(use_only_start_and_end_){
            if(i != 0 && i != (trained_template_vec_.size() - 1)){
                continue;
            }
        }
        if(trained_template_vec_.at(i).non_zero_area_templates.size() < 1){
            lw << "[FAIL] No template is detected"
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
            t_execute = 0;
            return 0;
        }
    }
    if(source_img.empty()){
        lw << "[FAIL] No source image is detected"
           << ", File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;
        t_execute = 0;
        return 0;
    }

    if(dm::log){
        hipDeviceSynchronize();
    }
    struct timespec start_all;//clock_t start_all = clock();
    clock_gettime(CLOCK_MONOTONIC, &start_all);
    struct timespec tp_start, tp_end;
    double host_elapsed;
    clock_gettime(CLOCK_MONOTONIC, &tp_start);
    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }

    orig_source_img_mat_ = source_img.clone();

    if(dm::log){
    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
       << ", in " << __func__ << lw.endl;
    }
    // Paras. setting #FIXME
    float angle_interval_fine_align = 1.0;
    float angle_interval_coarse_align = 5.0;
    float angle_min = -45.0;
    float angle_max = 45.0;
    int num_angles;
    int num_pyramid = 1;
    rslt_match.clear();

    std::string params_path = this->params_path_ + parameter_matching_file;
    if (!fio::exists(params_path)){
        if(dm::log){
            std::cerr << "File isn't opened." << std::endl;
        }
        CreateParamsIniFile(params_path);
    }else {
        // FileIO. 20220419. Jimmy. #fio14
        fio::FileIO inim(params_path, fio::FileioFormat::INI);
        inim.IniSetSection("ParameterChamfer");
        angle_min = inim.IniReadtoFloat("angle_min");
        angle_max = inim.IniReadtoFloat("angle_max");
        angle_interval_coarse_align = inim.IniReadtoFloat("angle_coarse_interval");
        angle_interval_fine_align = inim.IniReadtoFloat("angle_fine_interval");

        inim.close();
    }
    num_angles = (angle_max - angle_min ) / angle_interval_fine_align;


    assert(angle_min < angle_max);
    assert(angle_min >= -180);
    assert(angle_max <= 180);
    assert(angle_interval_fine_align <= angle_interval_coarse_align);
    assert(angle_interval_coarse_align > 0 && angle_interval_coarse_align <= 25);
    assert(angle_interval_fine_align > 0 && angle_interval_fine_align <= 10);

    // Template
    short num_pyrmd = trained_template_vec_.size(); // 1, 2, 4, 8, ...
    pre_proc_src_vec_.resize(num_pyrmd);

    // Original source image
    int orig_src_width = orig_source_img_mat_.cols;
    int orig_src_height = orig_source_img_mat_.rows;
    int orig_tmp_width = orig_template_img_mat_.cols;
    int orig_tmp_height = orig_template_img_mat_.rows;
    short padding_w = orig_tmp_width / 2;
    short padding_h = orig_tmp_height / 2;

    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    if(dm::log){
        checkCUDAandSysInfo();
        hipDeviceSynchronize();
    }
    if(dm::log){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
            << ", in " << __func__ << lw.endl;
    }
    image_proc->SkipBlurContour(false, false);
    orig_source_contours_mat_.release();

    CreateSrcNonZeroMat2(orig_source_img_mat_, pre_proc_src_vec_,
                        num_pyrmd,
                        orig_tmp_width, orig_tmp_height, orig_path);

    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &tp_end);
    host_elapsed = clock_diff (&tp_start, &tp_end);
    clock_gettime(CLOCK_MONOTONIC, &tp_start);
    lw << "[DEBUG] Time elapsed - TOTAL (CreateSrcNonZeroMat): " << host_elapsed << "s" << lw.endl;


    int num_result_scores;
    int orig_tot_mem_size;
    std::shared_ptr<float[]> score_candidates_result;
    std::shared_ptr<int[]> score_candidates_result_index;

    std::vector<cv::Point> non_zero_filtered_cand;

    TrainTemplatePtr* trained_template = nullptr;
    bool is_last = false;
    for (int pyr = 0; pyr < num_pyrmd; ++pyr) {
        if(use_only_start_and_end_){
            if(pyr != 0 && pyr != (num_pyrmd - 1)){
                continue;
            }
        }

        // Index of pyramid scale
        // From the last
        int template_pyrmd_index = num_pyrmd - 1 - pyr;
        PreprocessedSourcePtr& pre_proc_src = pre_proc_src_vec_.at(template_pyrmd_index);
        trained_template = &trained_template_vec_.at(template_pyrmd_index);

        if(dm::log){
            lw << "[DEBUG] File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
        }
        // Check if the nonzero parts are detected
        if(pre_proc_src.length_source_candidate < 1){
            t_execute = 0;
            return 0;
        }
        short resize_scale = std::pow(2, template_pyrmd_index);
        int re_src_width = pre_proc_src.width;
        int re_src_height = pre_proc_src.height;
        int re_tmp_width = orig_tmp_width / resize_scale;
        int re_tmp_height = orig_tmp_height / resize_scale;
        short padding_w = re_tmp_width / 2;
        short padding_h = re_tmp_height / 2;



        if(dm::log){
            lw << "[DEBUG] ====== template_pyrmd_index: " << template_pyrmd_index << " ======" << lw.endl;
            lw << "[DEBUG] length_source_candidate: " << pre_proc_src.length_source_candidate << lw.endl;
            lw << "[DEBUG] num_templates: " << trained_template->num_templates << lw.endl;
            lw << "[DEBUG] total_memory_size: " << trained_template->total_memory_size << lw.endl;
        }

        ////// CUDA memory
        float* cuda_flat_source_dist_transfrom = nullptr;
        // Candidate
        short* cuda_source_candidate_col = nullptr;
        short* cuda_source_candidate_row = nullptr;
        // Score
        float* cuda_flat_score_map = nullptr;
        int* cuda_flat_score_map_index = nullptr;
        // Template non-zero
        short* cuda_flat_template_non_zero_col = nullptr;
        short* cuda_flat_template_non_zero_row = nullptr;
        // Weight for scoring
        float* cuda_flat_template_weight = nullptr;

        // The number of non-zero in each template.
        int* cuda_template_num_non_zero = nullptr;
        // Used in the sorting process.
        float* cuda_score_candidates = nullptr;
        int* cuda_score_candidates_index = nullptr;

        ////// Memory
        // Candidate
        short* source_candidate_col = pre_proc_src.source_candidate_col.get();
        short* source_candidate_row = pre_proc_src.source_candidate_row.get();
        // Template non-zero
        short* flat_template_non_zero_col = trained_template->flat_template_non_zero_col_.get();
        short* flat_template_non_zero_row = trained_template->flat_template_non_zero_row_.get();
        // Weight for scoring
        float* flat_template_weight = trained_template->flat_template_weight_.get();
        // The number of non-zero in each template.
        int* template_num_non_zero = trained_template->num_non_zeros.get();


        ////////////////////////////////// Memory Trans. {Host to Device} //////////////////////////////////
        size_t re_src_size = re_src_width * re_src_height; // Including expanding(padding) case
        // Distance Transform (Original size)
        hipMalloc(&cuda_flat_source_dist_transfrom, re_src_size * sizeof(float));
        hipMemcpy(cuda_flat_source_dist_transfrom, pre_proc_src.flat_source_dist_transfrom,
                   re_src_size * sizeof(float), hipMemcpyHostToDevice);

        size_t cand_size = pre_proc_src.length_source_candidate;
        // Candidate
        hipMalloc(&cuda_source_candidate_col, cand_size * sizeof(short));
        hipMemcpy(cuda_source_candidate_col, source_candidate_col,
                   cand_size * sizeof(short), hipMemcpyHostToDevice);
        hipMalloc(&cuda_source_candidate_row, cand_size * sizeof(short));
        hipMemcpy(cuda_source_candidate_row, source_candidate_row,
                   cand_size * sizeof(short), hipMemcpyHostToDevice);

        // Score
        /// No need to make entire score map. Because the only non-zero positions will be computed.
        /// (Reducing the resource and process time.)
        /// the score size, started from the minimum pyramid scale
        int score_mem_size = trained_template->num_templates
                                * cand_size;
        hipMalloc(&cuda_flat_score_map, score_mem_size * sizeof(float));
        hipMalloc(&cuda_flat_score_map_index, score_mem_size * sizeof(int));

        size_t temp_nonzero_size = trained_template->total_memory_size;  // Including rotated templates
        // Template non-zero
        hipMalloc(&cuda_flat_template_non_zero_col, temp_nonzero_size * sizeof(short));
        hipMalloc(&cuda_flat_template_non_zero_row, temp_nonzero_size * sizeof(short));
        hipMemcpy(cuda_flat_template_non_zero_col, flat_template_non_zero_col,
                   temp_nonzero_size * sizeof(short), hipMemcpyHostToDevice);
        hipMemcpy(cuda_flat_template_non_zero_row, flat_template_non_zero_row,
                   temp_nonzero_size * sizeof(short), hipMemcpyHostToDevice);

        // Weight for scoring
        hipMalloc(&cuda_flat_template_weight, temp_nonzero_size * sizeof(float));
        hipMemcpy(cuda_flat_template_weight, flat_template_weight,
                   temp_nonzero_size * sizeof(float), hipMemcpyHostToDevice);



        // The number of non-zero in each template.
        hipMalloc(&cuda_template_num_non_zero, trained_template->num_templates * sizeof(int));
        hipMemcpy(cuda_template_num_non_zero, template_num_non_zero,
                   trained_template->num_templates * sizeof(int), hipMemcpyHostToDevice);

        //////////////////////////////////////////////////////////////////////////////////////////////////////


        ////////////////////////////////// CUDA: Set the CUDA memory block //////////////////////////////////
        int block_size = 32;    // max: 32
        dim3 threads_in_a_block(block_size, block_size);    // 16x16 threads
        int num_blocks_w = (int)((pre_proc_src.source_num_candidate_width + block_size - 1) / block_size);
        int num_blocks_h = (int)((pre_proc_src.source_num_candidate_height + block_size - 1) / block_size);
        dim3 num_blocks(num_blocks_w, num_blocks_h);

        // short length_source_candidate, short source_num_candidate_width, short source_num_candidate_height,
        // these should be computed for threads....
        // get total length
        // separate them
        hipError_t cuda_error = hipGetLastError();
        if(cuda_error != hipError_t::hipSuccess){
            lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
        }

        lw << "[DEBUG] length_source_candidate: " << pre_proc_src.length_source_candidate << lw.endl;
        lw << "[DEBUG] source_num_candidate_width: " << pre_proc_src.source_num_candidate_width << lw.endl;
        lw << "[DEBUG] source_num_candidate_height: " << pre_proc_src.source_num_candidate_height << lw.endl;
        lw << "[DEBUG] source_candidate_col: " << source_candidate_col[100] << lw.endl;
        lw << "[DEBUG] source_candidate_row: " << source_candidate_row[100] << lw.endl;
        lw << "[DEBUG] flat_template_non_zero_col: " << flat_template_non_zero_col[100] << lw.endl;
        lw << "[DEBUG] flat_template_non_zero_row: " << flat_template_non_zero_row[100] << lw.endl;
        lw << "[DEBUG] temp_nonzero_size: " << temp_nonzero_size<< lw.endl;
        lw << "[DEBUG] flat_template_weight: " << flat_template_weight[100] << lw.endl;

        ChamferMatch2<<<num_blocks, threads_in_a_block>>>(cuda_flat_source_dist_transfrom,
                                     cuda_source_candidate_col, cuda_source_candidate_row,
                                     re_src_width, re_src_height,
                                     trained_template->num_templates,
                                     cuda_flat_template_non_zero_col, cuda_flat_template_non_zero_row, cuda_template_num_non_zero,
                                     cuda_flat_template_weight, (short)use_weight,
                                     pre_proc_src.length_source_candidate,
                                     pre_proc_src.source_num_candidate_width,
                                     pre_proc_src.source_num_candidate_height,
                                     cuda_flat_score_map, cuda_flat_score_map_index);
        if(dm::log){
            hipDeviceSynchronize();
            checkCUDAandSysInfo();
        }
        cuda_error = hipGetLastError();
        if(cuda_error != hipError_t::hipSuccess){
            lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
        }
        if(dm::log){
            hipDeviceSynchronize();
            clock_gettime(CLOCK_MONOTONIC, &tp_end);
            host_elapsed = clock_diff (&tp_start, &tp_end);
            clock_gettime(CLOCK_MONOTONIC, &tp_start);
            lw << "[DEBUG] Time elapsed - ChamferMatch: " << host_elapsed << "s" << lw.endl;
        }
        lw << "[DEBUG] num_scores(num_templates x candidates): " << score_mem_size << lw.endl;

        //////// TEST ////////
        if(dm::log){
            // ex) orig_tot_mem_size = 40
            std::shared_ptr<float[]> TEST_score_candidates_result(new float[score_mem_size]());
            std::shared_ptr<int[]> TEST_score_candidates_result_index(new int[score_mem_size]());

            hipMemcpy(TEST_score_candidates_result.get(), cuda_flat_score_map, score_mem_size * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(TEST_score_candidates_result_index.get(), cuda_flat_score_map_index, score_mem_size * sizeof(int), hipMemcpyDeviceToHost);
            for(int i = 0; i < 100; i ++){
                if(TEST_score_candidates_result[i] < 0.1){
                    lw << "[DEBUG] Score: " << TEST_score_candidates_result[i] << ", " << TEST_score_candidates_result_index[i] << lw.endl;
                }

            }
            hipDeviceSynchronize();
            cuda_error = hipGetLastError();
            if(cuda_error != hipError_t::hipSuccess){
                lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
                   << ", File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
            }


            if(dm::match_img){
                try{
                    cv::Mat color_img;
                    if(source_img.channels() == 1){
                        cv::cvtColor(source_img, color_img, cv::COLOR_GRAY2BGR);
                    }
                    else {
                        color_img = source_img.clone();
                    }
                    cv::resize(color_img, color_img, cv::Size(source_img.cols / resize_scale, source_img.rows / resize_scale));
                    cv::Mat result_mat_img = cv::Mat::zeros(re_src_height, re_src_width, CV_8UC3);
                    cv::Rect crop_roi = cv::Rect(padding_w, padding_h, color_img.cols, color_img.rows);
                    color_img.copyTo(result_mat_img(crop_roi));

                    if(dm::log){
                        lw << "[DEBUG] resize_scale: " << resize_scale << lw.endl;
                        lw << "[DEBUG] padding_w: " << padding_w << lw.endl;
                        lw << "[DEBUG] padding_h: " << padding_h << lw.endl;
                        lw << "[DEBUG] re_src_height: " << re_src_height << lw.endl;
                        lw << "[DEBUG] re_src_width: " << re_src_width << lw.endl;
                        lw << "[DEBUG] color_img.cols: " << color_img.cols << lw.endl;
                        lw << "[DEBUG] color_img.rows: " << color_img.rows << lw.endl;
                    }

                    for(int i = 0; i < score_mem_size; i ++){
                        int source_target = TEST_score_candidates_result_index[i] % pre_proc_src.length_source_candidate;
                        int source_target_rot = TEST_score_candidates_result_index[i] / pre_proc_src.length_source_candidate;
                        short x__= pre_proc_src.source_candidate_col[source_target];
                        short y__ = pre_proc_src.source_candidate_row[source_target];

//                        result_mat_img.at<uchar>(y__, x__) = 255;

                        cv::Vec3b& vec3b = result_mat_img.at<cv::Vec3b>(y__, x__) ;
                        vec3b[0] = 0;
                        vec3b[1] = 0;
                        vec3b[2] = 255;
                    }
                    // To show original candidates before expanding
                    for(int i = 0; i < non_zero_filtered_cand.size(); i ++){
                        short x__= non_zero_filtered_cand.at(i).x;
                        short y__ = non_zero_filtered_cand.at(i).y;

                        cv::Vec3b& vec3b = result_mat_img.at<cv::Vec3b>(y__, x__) ;
                        vec3b[0] = 0;
                        vec3b[1] = 255;
                        vec3b[2] = 255;
                    }

                    cv::imwrite(data_path + "results/0_candidate_" + std::to_string(pyr) + ".png", result_mat_img);
                }catch(cv::Exception e){
                    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                    std::string error_what = e.what();
                    lw << error_what << lw.endl;
                }catch(...){
                    lw << "Error. " << "File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                }
            }
            TEST_score_candidates_result.reset();
            TEST_score_candidates_result_index.reset();

            if(dm::log){
                hipDeviceSynchronize();
                clock_gettime(CLOCK_MONOTONIC, &tp_end);
                host_elapsed = clock_diff (&tp_start, &tp_end);
                clock_gettime(CLOCK_MONOTONIC, &tp_start);
                lw << "[DEBUG] Time elapsed - Test (Visualization for candidates): " << host_elapsed << "s" << lw.endl;
            }

        }
        //////// END: TEST ////////

        // Do score sorting
        int num_scores = score_mem_size;    // the number of templates * length_source_candidate

        is_last = (pyr == (num_pyrmd - 1));
        bool is_fine_detection = false;
        int steps = 3;
        std::vector<int> target_num_score_list; // if a target_num_score is 10, one thread sort out 10 sorted values.
        std::vector<int> divider_list;
        target_num_score_list.resize(steps);
        divider_list.resize(steps);


        double init_pow; // 1/2, 2/3, 4/3, ...
        if(pyr < (num_pyrmd - 2)){
            init_pow = (double)num_pyrmd / (double)(2 + num_pyrmd);  // 1/2, 2/3, 4/3, ...
        }
        else{
            init_pow = (double)num_pyrmd / (double)(4 + num_pyrmd);  // 1/2, 2/3, 4/3, ...
        }
        double pow_step = (1.0 - init_pow) / (double)(steps);
//        int final_list_num = std::pow(num_scores, init_pow);
//        int step_unit = num_scores - final_list_num;
//        double step_unit_pow = 1.0 / (double)(steps);
//        step_unit = std::pow(step_unit, step_unit_pow);

        if(dm::log){
            lw << "[DEBUG] ======================================================================" << lw.endl;
            lw << "[DEBUG] init_pow: " << init_pow << lw.endl;
            lw << "[DEBUG] pow_step: " << pow_step << lw.endl;
//            lw << "[DEBUG] final_list_num: " << final_list_num << lw.endl;
//            lw << "[DEBUG] step_unit: " << step_unit << lw.endl;
//            lw << "[DEBUG] step_unit_pow: " << step_unit_pow << lw.endl;
        }

//        for (int s = 0; s < steps; ++s) {
//            target_num_score_list.at(s) = 10;
//            int total = std::pow(num_scores, 1.0 - (pow_step * ((double)s + 1.0)));
//            if(total <= 10){
//                target_num_score_list.at(s) = total;
//                if(total <= 0){
//                    total = std::pow(num_scores, 1.0 - ((pow_step / 2) * (double)s + 1.0));
//                }
//                divider_list.at(s) = 1;
//            }
//            else{
//                divider_list.at(s) = total / 10;
//            }

//            if(s == (steps - 1)){
//                if(total <= 10){
//                    target_num_score_list.at(s) = total;
//                }
//                divider_list.at(s) = 1;
//            }
//            if(is_last){
//                is_fine_detection = true;
//            }
//            if(dm::log){
//                lw << "[DEBUG] divider_list.at(s): " << divider_list.at(s) << lw.endl;
//                lw << "[DEBUG] target_num_score_list.at(s): " << target_num_score_list.at(s) << lw.endl;
//            }
//        }

//        num_result_scores = target_num_score_list.at(target_num_score_list.size() - 1)
//                            * divider_list.at(divider_list.size() - 1);

        num_result_scores = 20;

        if(dm::log){
            hipDeviceSynchronize();
            clock_gettime(CLOCK_MONOTONIC, &tp_end);
            host_elapsed = clock_diff (&tp_start, &tp_end);
            clock_gettime(CLOCK_MONOTONIC, &tp_start);
            lw << "[DEBUG] Time elapsed - ETC: " << host_elapsed << "s" << lw.endl;
            lw << "[DEBUG] num_result_scores: " << num_result_scores << lw.endl;
        }
        /// Sort using CUB library
        std::shared_ptr<float[]> score_candidates(new float[num_result_scores]());
        std::shared_ptr<int[]> score_candidates_index(new int[num_result_scores]());
        device_wise_sort(cuda_flat_score_map, cuda_flat_score_map_index, score_mem_size,
                         score_candidates.get(), score_candidates_index.get(), num_result_scores);
        if(dm::log){
            hipDeviceSynchronize();
            clock_gettime(CLOCK_MONOTONIC, &tp_end);
            host_elapsed = clock_diff (&tp_start, &tp_end);
            clock_gettime(CLOCK_MONOTONIC, &tp_start);
            lw << "[DEBUG] Time elapsed - Sorting (CUB_DEVICE_WISE): " << host_elapsed << "s" << lw.endl;
        }
        ///

//        hipMalloc(&cuda_score_candidates, num_result_scores * sizeof(float));
//        hipMalloc(&cuda_score_candidates_index, num_result_scores * sizeof(int));
//        SortScores(cuda_flat_score_map, cuda_flat_score_map_index,
//                    cuda_score_candidates, cuda_score_candidates_index,
//                    num_scores, steps,
//                   target_num_score_list, divider_list,
//                    is_fine_detection);

//        if(dm::log){
//            hipDeviceSynchronize();
//            clock_gettime(CLOCK_MONOTONIC, &tp_end);
//            host_elapsed = clock_diff (&tp_start, &tp_end);
//            clock_gettime(CLOCK_MONOTONIC, &tp_start);
//            lw << "[DEBUG] Time elapsed - Sorting: " << host_elapsed << "s" << lw.endl;
//        }

//        hipMemcpy(score_candidates.get(), cuda_score_candidates, num_result_scores * sizeof(float), hipMemcpyDeviceToHost);
//        hipMemcpy(score_candidates_index.get(), cuda_score_candidates_index, num_result_scores * sizeof(int), hipMemcpyDeviceToHost);

//        FreeCudaMemory(&cuda_score_candidates);
//        FreeCudaMemory(&cuda_score_candidates_index);



        if(is_last){
            for (unsigned int i = 0; i < num_result_scores; i++){
                int source_target = score_candidates_index[i] % pre_proc_src.length_source_candidate;
                int source_target_rot = score_candidates_index[i] / pre_proc_src.length_source_candidate;
                short x__= pre_proc_src.source_candidate_col[source_target];
                short y__ = pre_proc_src.source_candidate_row[source_target];
                lw << "[DEBUG] Score: " << score_candidates[i] << lw.endl;
                lw << "[DEBUG] source_target: " << source_target << lw.endl;
                lw << "[DEBUG] source_target_rot: " << source_target_rot << lw.endl;
                lw << "[DEBUG] x__: " << x__ << lw.endl;
                lw << "[DEBUG] y__: " << y__ << lw.endl;
            }

            score_candidates_result = score_candidates;
            score_candidates_result_index = score_candidates_index;
        }


        ////////////////////////// Free CUDA memory //////////////////////////
        template_num_non_zero = NULL;
        // Free memory
        FreeCudaMemory(&cuda_flat_template_non_zero_col);
        FreeCudaMemory(&cuda_flat_template_non_zero_row);
        FreeCudaMemory(&cuda_flat_template_weight);
        FreeCudaMemory(&cuda_source_candidate_col);
        FreeCudaMemory(&cuda_source_candidate_row);
        FreeCudaMemory(&cuda_template_num_non_zero);
        ////////////////////////// END: Free CUDA memory //////////////////////////


        if(pyr != (num_pyrmd - 1)){
            ////////////////////////// Prepare for the next step //////////////////////////
            int next_pyrmd_index = template_pyrmd_index - 1;
            unsigned int count = 0;
            int pyrmd_unit = 2;
            if(use_only_start_and_end_){
                if(pyr != 0){
                    continue;
                }
                next_pyrmd_index = 0;
                pyrmd_unit = resize_scale;
            }

            PreprocessedSourcePtr& pre_proc_src_next = pre_proc_src_vec_.at(next_pyrmd_index);
            // Generate the next candidates
            std::vector<cv::Point> filter_target(num_result_scores);
            bool do_pass = false;

            if(dm::log){
                hipDeviceSynchronize();
                clock_gettime(CLOCK_MONOTONIC, &tp_end);
                host_elapsed = clock_diff (&tp_start, &tp_end);
                clock_gettime(CLOCK_MONOTONIC, &tp_start);
                lw << "[DEBUG] Time elapsed - ETC: " << host_elapsed << "s" << lw.endl;
            }

            bool orig_dilate_method = false;
            if(orig_dilate_method){
                for (unsigned int i = 0; i < num_result_scores; i++){
                    int source_target = score_candidates_index[i] % pre_proc_src.length_source_candidate;
                    short fitx = pre_proc_src.source_candidate_col[source_target] * pyrmd_unit;
                    short fity = pre_proc_src.source_candidate_row[source_target] * pyrmd_unit;

                    do_pass = false;
                    for (int j = i - 1; j >= 0; j--){
                        if(filter_target.at(j).x == fitx && filter_target.at(j).y == fity){
            //                lw << "[DEBUG] dupl: " << (int)fitx << ", " << (int)fity << lw.endl;
                            do_pass = true;
                            break;
                        }
                    }
                    if(!do_pass){
                        filter_target.at(i).x = fitx;
                        filter_target.at(i).y = fity;
                        count++;
                    }
                    else{
                        filter_target.at(i).x = 0;
                        filter_target.at(i).y = 0;
                    }
                }
                if(count == 0){
                    lw << "[FAIL] No template is detected"
                       << ", File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                    t_execute = 0;
                    return 0;
                }
                if(dm::log){
                    hipDeviceSynchronize();
                    clock_gettime(CLOCK_MONOTONIC, &tp_end);
                    host_elapsed = clock_diff (&tp_start, &tp_end);
                    clock_gettime(CLOCK_MONOTONIC, &tp_start);
                    lw << "[DEBUG] Time elapsed - Filter Candidate: " << host_elapsed << "s" << lw.endl;
                }




                count = 0;
                std::vector<cv::Point> filter_expanded_target;
                for (int i = 0; i < num_result_scores; i++){
                    short fitx = filter_target.at(i).x;
                    short fity = filter_target.at(i).y;
                    if(fitx == 0 && fity == 0) continue;

                    for (short upx = 0; upx < pyrmd_unit*2; upx++){      // #FIXME: upx++ => upx+=pow(2, pyrNum-i)
                        for (short upy = 0; upy < pyrmd_unit*2; upy++){
                            short dilated_x = fitx + upx - pyrmd_unit;
                            short dilated_y = fity + upy - pyrmd_unit;
                            do_pass = false;
                            for (int j = count - 1; j >= 0; j--){
                                if(filter_expanded_target.at(j).x == dilated_x
                                        && filter_expanded_target.at(j).y == dilated_y){
                                    do_pass = true;
                                    break;
                                }
                            }
                            if(!do_pass){
                                if(dilated_x < 0 ) dilated_x = 0;
                                if(dilated_y < 0 ) dilated_y = 0;
                                filter_expanded_target.push_back(cv::Point(dilated_x, dilated_y));
                                count++;
                            }
                        }
                    }
                }
                orig_tot_mem_size = filter_expanded_target.size();
                MakeSharedPtr(pre_proc_src_next.source_candidate_col, orig_tot_mem_size);
                MakeSharedPtr(pre_proc_src_next.source_candidate_row, orig_tot_mem_size);
                for (int i = 0; i < orig_tot_mem_size; i++){
                    pre_proc_src_next.source_candidate_col[i] = filter_expanded_target.at(i).x;
                    pre_proc_src_next.source_candidate_row[i] = filter_expanded_target.at(i).y;
                }
                pre_proc_src.source_candidate_col.reset();
                pre_proc_src.source_candidate_row.reset();
                if(dm::log){
                    lw << "[DEBUG] orig_tot_mem_size: " << orig_tot_mem_size << lw.endl;
                    lw << "[DEBUG] Filtered count: " << (int)count << lw.endl;
                }

                filter_expanded_target.clear();
                if(dm::log){
                    hipDeviceSynchronize();
                    clock_gettime(CLOCK_MONOTONIC, &tp_end);
                    host_elapsed = clock_diff (&tp_start, &tp_end);
                    clock_gettime(CLOCK_MONOTONIC, &tp_start);
                    lw << "[DEBUG] Time elapsed - Expand candidate using original method: " << host_elapsed << "s" << lw.endl;
                }



                filter_target.clear();
                if(dm::log){
                    lw << "[DEBUG] Generated count: " << (int)count << lw.endl;
                }
            }
            else{
                //// Dilating using OpenCV
                /// Slower than original method. Approx. 8ms, but original is about 2ms.
                cv::Mat filtered_candidate = cv::Mat::zeros(cv::Size(pre_proc_src_next.width, pre_proc_src_next.height), CV_8UC1);
                cv::Mat dilated_candidate;
                for (int i = 0; i < num_result_scores; i++){
                    int source_target = score_candidates_index[i] % pre_proc_src.length_source_candidate;
                    short fitx = pre_proc_src.source_candidate_col[source_target];
                    short fity = pre_proc_src.source_candidate_row[source_target];

                    // Jang 20220713
                    for (short upx = 0; upx < 3; upx++){
                        for (short upy = 0; upy < 3; upy++){
                            short dilated_fitx = (fitx + upx - 1) * pyrmd_unit;
                            short dilated_fity = (fity + upy - 1) * pyrmd_unit;
                            if(dilated_fitx < 0 || dilated_fity < 0) continue;
                            filtered_candidate.at<uchar>(dilated_fity, dilated_fitx) = 255;
                        }
                    }

                }
                if(dm::match_img){
                    cv::findNonZero(filtered_candidate, non_zero_filtered_cand);
                }
                cv::morphologyEx(filtered_candidate, dilated_candidate, cv::MORPH_DILATE,
                                cv::Mat::ones(cv::Size(pyrmd_unit * 3, pyrmd_unit * 3) , CV_8UC1));
                std::vector<cv::Point> non_zero_;
                cv::findNonZero(dilated_candidate, non_zero_);
                orig_tot_mem_size = non_zero_.size();
                MakeSharedPtr(pre_proc_src_next.source_candidate_col, orig_tot_mem_size);
                MakeSharedPtr(pre_proc_src_next.source_candidate_row, orig_tot_mem_size);
                pre_proc_src.source_candidate_col.reset();
                pre_proc_src.source_candidate_row.reset();
                for (int i = 0; i < orig_tot_mem_size; i++){
                    pre_proc_src_next.source_candidate_col[i] = non_zero_.at(i).x;
                    pre_proc_src_next.source_candidate_row[i] = non_zero_.at(i).y;
                }

                if(dm::log){
                    hipDeviceSynchronize();
                    clock_gettime(CLOCK_MONOTONIC, &tp_end);
                    host_elapsed = clock_diff (&tp_start, &tp_end);
                    clock_gettime(CLOCK_MONOTONIC, &tp_start);
                    lw << "[DEBUG] Time elapsed - Expand candidate using OpenCV: " << host_elapsed << "s" << lw.endl;
                }
            }

            pre_proc_src_next.length_source_candidate = orig_tot_mem_size;
            pre_proc_src_next.source_num_candidate_width = (short)sqrt(pre_proc_src_next.length_source_candidate) + 1;
            pre_proc_src_next.source_num_candidate_height = pre_proc_src_next.source_num_candidate_width;

            score_candidates.reset();
            score_candidates_index.reset();

            if(dm::log){
                hipDeviceSynchronize();
                clock_gettime(CLOCK_MONOTONIC, &tp_end);
                host_elapsed = clock_diff (&tp_start, &tp_end);
                clock_gettime(CLOCK_MONOTONIC, &tp_start);
                lw << "[DEBUG] Time elapsed - Prepare for the next step: " << host_elapsed << "s" << lw.endl;
            }
            ////////////////////////// END: Prepare for the next step //////////////////////////
        }
        ////////////////////////// Free CUDA memory //////////////////////////
        FreeCudaMemory(&cuda_flat_source_dist_transfrom);
        ////////////////////////// END: Free CUDA memory //////////////////////////
    }

    PreprocessedSourcePtr& pre_proc_src = pre_proc_src_vec_.at(0);

    int best_case = 0;
    double best_ncc = 0;
    cv::Mat copied_source_img = source_img.clone();
    int length_source_candidate = orig_tot_mem_size;
    int template_width = orig_tmp_width;
    int template_height = orig_tmp_height;
    short* source_candidate_col = pre_proc_src.source_candidate_col.get();
    short* source_candidate_row = pre_proc_src.source_candidate_row.get();
    cv::Mat source_canny_img;
    cv::Mat source_gray_img = copied_source_img.clone();
    cv::Mat rot_template_mat;
    std::vector<float> ncc_results_vec;

    ////////////////////////// NCC Score //////////////////////////
    try{
        // NCC computation preparation
        if(copied_source_img.channels() > 1)
            cv::cvtColor(copied_source_img, source_gray_img, cv::COLOR_BGR2GRAY);
        if(expand_img_){
            std::vector<int> expanded_xy_index;
            bool pass_compute = false;
            for(int j = 0; j < num_result_scores; j++){
                int source_target = score_candidates_result_index[j] % length_source_candidate;
                pass_compute = false;
                for(int k = 0; k < expanded_xy_index.size(); k ++){
                    if(source_target == expanded_xy_index.at(k)){
                        pass_compute = true;
                    }
                }
                if(pass_compute){
                    continue;
                }

                expanded_xy_index.push_back(source_target);

                if(dm::log){
                    std::cout << "=================orig===========================" << std::endl;
                    std::cout << "source_target: " << source_target << std::endl;
                    std::cout << "score_candidates_result_index[j]: " << score_candidates_result_index[j] << std::endl;
                    std::cout << "source_candidate_col[source_target]: " << source_candidate_col[source_target] << std::endl;
                    std::cout << "source_candidate_row[source_target]: " << source_candidate_row[source_target] << std::endl;
                    std::cout << "============================================" << std::endl;
                }
                source_candidate_col[source_target] -= (short)(template_width / 2);
                source_candidate_row[source_target] -= (short)(template_height / 2);
                if(dm::log){
                    std::cout << "source_candidate_col[source_target]: " << source_candidate_col[source_target] << std::endl;
                    std::cout << "source_candidate_row[source_target]: " << source_candidate_row[source_target] << std::endl;
                }
            }
        }

        // Get the best case
        fs::create_directories(data_path + "results/");
        for(int j = 0; j < num_result_scores; j++){
            int template_scale_rot = score_candidates_result_index[j] / length_source_candidate;
            int template_scale = template_scale_rot / num_angles;
            int template_rot = template_scale_rot % num_angles; // + angle_min; // Jang 20211203    // This is the position of rotation from the beginning. (ex) from angle_min.
            int source_target = score_candidates_result_index[j] % length_source_candidate;

//            lw << "Index: " << score_candidates_result_index_orig[j] << ", Score: " << score_candidates_result_orig[j] << lw.endl;
//            lw << "[ " << source_candidate_col[source_target] << ", " << source_candidate_row[source_target] << " ]" << lw.endl;
//            lw << "Scale: " << template_scale << ", Rot: " << template_rot << lw.endl;

            // Mask
            cv::Rect crop_roi;
            cv::Rect crop_temp_roi;

            int crop_x;
            int crop_y;
            int crop_w;
            int crop_h;
            int crop_tmp_x;
            int crop_tmp_y;
            crop_x = source_candidate_col[source_target];
            crop_y = source_candidate_row[source_target];
            crop_w = template_width;
            crop_h = template_height;
            crop_tmp_x = 0;
            crop_tmp_y = 0;
            if(source_candidate_col[source_target] < 0){
                crop_x = 0;
                crop_w = template_width + source_candidate_col[source_target];
                crop_tmp_x = -source_candidate_col[source_target];
            }
            if(source_candidate_row[source_target] < 0){
                crop_y = 0;
                crop_h = template_height + source_candidate_row[source_target];
                crop_tmp_y = -source_candidate_row[source_target];
            }
            if((source_candidate_col[source_target] + template_width) >  orig_source_img_mat_.cols){
                crop_w = (orig_source_img_mat_.cols - source_candidate_col[source_target]);
            }
            if((source_candidate_row[source_target] + template_height) >  orig_source_img_mat_.rows){
                crop_h = (orig_source_img_mat_.rows - source_candidate_row[source_target]);
            }
            crop_roi = cv::Rect(crop_x, crop_y, crop_w, crop_h);
            crop_temp_roi = cv::Rect(crop_tmp_x, crop_tmp_y, crop_w, crop_h);


            cv::Size temp_resized_size;
            temp_resized_size.width = crop_w;
            temp_resized_size.height = crop_h;
            if(trained_template->templates_gray.size() > template_scale_rot){
                // NCC computation
//                rot_template_mat = trained_template->templates_gray.at(template_scale_rot)(crop_temp_roi);
                rot_template_mat = rot_template_mat.zeros(temp_resized_size, trained_template->templates_gray.at(template_scale_rot).type());
                trained_template->templates_gray.at(template_scale_rot)(crop_temp_roi).copyTo(rot_template_mat);
                // Average of source
                double avg_source;
                avg_source = cv::mean(source_gray_img(crop_roi))[0];

                // Average of template
                double avg_template;
                avg_template = cv::mean(rot_template_mat)[0];
                if(dm::match_img){
                    cv::imwrite(data_path + "results/" + std::to_string(j) + "_cropped_source_gray_img.png", source_gray_img(crop_roi));
                    cv::imwrite(data_path + "results/" + std::to_string(j) + "_rot_template_mat.png", rot_template_mat);
                }
                // Sum of child values
                double sum_numerator = 0;
                double sum_denominator_0 = 0;
                double sum_denominator_1 = 0;
                for(int r = 0; r < rot_template_mat.rows; r ++){
                    for(int c = 0; c < rot_template_mat.cols; c++){
                        double compute_val_0 = rot_template_mat.at<uchar>(r, c) - avg_template;
                        double compute_val_1 = source_gray_img.at<uchar>(r + crop_roi.y, c + crop_roi.x) - avg_source;
                        sum_numerator += (compute_val_0 * compute_val_1);

                        sum_denominator_0 += std::pow(compute_val_0, 2);
                        sum_denominator_1 += std::pow(compute_val_1, 2);

                    }
                }

                sum_denominator_0 = std::sqrt(sum_denominator_0);
                sum_denominator_1 = std::sqrt(sum_denominator_1);



                double ncc_result = 0;
                ncc_result = sum_numerator / (sum_denominator_0 * sum_denominator_1);
//                ncc_result = std::fabs(ncc_result);

                if(dm::log){
                    lw << "avg_source: " << avg_source << lw.endl;
                    lw << "avg_template: " <<  avg_template << lw.endl;
                    lw << "sum_denominator_0: " << sum_denominator_0 << lw.endl;
                    lw << "sum_denominator_1: " << sum_denominator_1 << lw.endl;
                    lw << "Score(ncc_result): " << ncc_result << lw.endl;
                }

                ncc_results_vec.push_back(ncc_result);
                if(best_ncc < ncc_result){
                    best_ncc = ncc_result;
                    best_case = j;
                }

            }
        }
        lw << "best_case(ncc_result): " << best_case << lw.endl;

        if(dm::log){
            lw << "num_angles: " << num_angles << ", angle_min: " << angle_min << lw.endl;
        }
        {
            int j = best_case;
            int template_scale_rot = score_candidates_result_index[j] / length_source_candidate;
            int template_scale = template_scale_rot / num_angles;
            int template_rot = template_scale_rot % num_angles; // + angle_min; // Jang 20211203    // This is the position of rotation from the beginning. (ex) from angle_min.
            int source_target = score_candidates_result_index[j] % length_source_candidate;

            float detected_angle = ((float)template_rot * angle_interval_fine_align) + angle_min;

            float score_percentage;
            score_percentage = (4.0 - pow(score_candidates_result[j], 2)) / 4.0;
            if(score_percentage < 0) score_percentage = 0;

            lw << "------ Result ------" << lw.endl;
            lw << "Index: " << score_candidates_result_index[j] << ", Score: " << score_candidates_result[j] << ", NCC: " << best_ncc << lw.endl;
            lw << "[ " << source_candidate_col[source_target] << ", " << source_candidate_row[source_target] << " ]" << lw.endl;
            lw << "Scale: " << template_scale << ", Rot: " << template_rot << lw.endl;

            if(dm::match_img){
                // Mask
                cv::Rect crop_roi = cv::Rect(source_candidate_col[source_target] , source_candidate_row[source_target], template_width, template_height);

                cv::Mat mask_mat = trained_template->templates.at(template_scale_rot);
                lw << "--- template_scale_rot: " << template_scale_rot << lw.endl;
                lw << "mask_mat: " << mask_mat.cols << ", " << mask_mat.rows << lw.endl;
                cv::Mat color_mask = cv::Mat::zeros(mask_mat.size(), CV_8UC3);
                std::vector<cv::Point> non_zero_;
                cv::findNonZero(mask_mat, non_zero_);

                // Change pixels' color
                for(int p = 0; p < non_zero_.size(); p++){
                    cv::Vec3b& vec3b = color_mask.at<cv::Vec3b>(non_zero_.at(p).y, non_zero_.at(p).x) ;
                    vec3b[0] = 0;
                    vec3b[1] = 0;
                    vec3b[2] = 255;
                }

                lw << "File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
                lw << "color_mask: " << color_mask.cols << ", " << color_mask.rows << lw.endl;
                lw << "copied_source_img: " << copied_source_img.cols << ", " << copied_source_img.rows << lw.endl;
                lw << "mask_mat: " << mask_mat.cols << ", " << mask_mat.rows << lw.endl;
                lw << "crop_roi: " << crop_roi.width << ", " << crop_roi.height << lw.endl;

                try{
                    cv::cvtColor(copied_source_img, copied_source_img, cv::COLOR_GRAY2BGR);
                }
                catch(cv::Exception e){
                    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                    std::string error_what = e.what();
                    lw << error_what << lw.endl;
                }catch(...){
                    lw << "Error. " << "File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                }
                color_mask.copyTo(copied_source_img(crop_roi), mask_mat.clone());

                cv::imwrite(data_path + "results/" + std::to_string(test_count) + "_copied_source_img.png", copied_source_img);
                cv::imwrite(data_path + "results/" + std::to_string(test_count) + "_copied_mask_mat.png", color_mask);
                test_count++;
            }

            MatchData match_data( 0, 0, 1.0f, .0f, .0f );
            match_data.x = source_candidate_col[source_target] + template_width / 2;   // center position
            match_data.y = source_candidate_row[source_target] + template_height / 2;  // center position
            match_data.angle = detected_angle; // unit: degree
            match_data.coef = best_ncc; // score_candidates_result_orig[j];
            match_data.scale = template_scale;

            // If the matched position is too close to the image boundaries, the matching failed.
//            if(match_data.x < (template_width / 4) || match_data.y < (template_height / 4) ||
//                    orig_source_img_mat_.cols - match_data.x < (template_width / 4) ||
//                    orig_source_img_mat_.rows - match_data.y < (template_height / 4) ){
//                 throw(__LINE__);
//             }
            // Jang 20221024
            // Disabled the function.
            if(match_data.x < (template_width / 2) || match_data.y < (template_height / 2) ||
                   orig_source_img_mat_.cols - match_data.x < (template_width / 2) ||
                   orig_source_img_mat_.rows - match_data.y < (template_height / 2) ){
                throw(__LINE__);
            }

            rslt_match.push_back(match_data);

            // Save the best candidates into a file. (the same file which is used in the RST matching)
            FILE *fp_result_p=fopen((data_path + result_matched_area_path).c_str(), "wb");
            {
                float angle_ = (detected_angle / 180 * PI) + (PI / 2);
                float radius_ = template_height / 2;
                // x1, y1: center position
                // x2, y2: template's boundary position to show angle
                int x1, x2, y1, y2;
                x1 = source_candidate_col[source_target] + template_width / 2;
                y1 = source_candidate_row[source_target] + template_height / 2;

                x2 = x1 + (std::cos(angle_)*radius_)+0.5;
                y2 = y1 - (sin(angle_)*radius_)+0.5;

                cv::Point pp0{x1, y1};
                cv::Point pp1{x2, y2};
                cv::Point p1{
                        x1+(cos(angle_)*((template_height-1)/2))+(cos(angle_+ (PI / 2))*((template_width-1)/2)),
                        y1-(sin(angle_)*((template_height-1)/2))-(sin(angle_+ (PI / 2))*((template_width-1)/2))
                };
                cv::Point p2{
                        (p1.x+(cos(angle_- (PI / 2))*template_width)),
                        (p1.y-(sin(angle_- (PI / 2))*template_width))
                };
                cv::Point p3{
                        (p2.x+(cos(angle_- PI)*template_height)),
                        p2.y-(sin(angle_- PI)*template_height)
                };
                cv::Point p4{
                        p1.x+(cos(angle_+ PI)*template_height),
                        p1.y-(sin(angle_+ PI)*template_height)
                };

                fprintf(fp_result_p, "%d\n%d\n%d\n%d\n",pp0.x,pp0.y,pp1.x,pp1.y);
                fprintf(fp_result_p, "%d\n%d\n%d\n%d\n",p1.x,p1.y,p2.x,p2.y);
                fprintf(fp_result_p, "%d\n%d\n%d\n%d\n",p3.x,p3.y,p4.x,p4.y);
            }
            fclose(fp_result_p);    fp_result_p = NULL;
        }
    }
    catch(cv::Exception e){
        lw << "File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;
        std::string error_what = e.what();
        lw << error_what << lw.endl;
        rslt_match.resize(1);
        rslt_match.at(0).coef = 0;
    }
    catch(int line){
        rslt_match.resize(1);
        rslt_match.at(0).coef = 0;
    }
    catch(...){
        lw << "Error. " << "File " <<  __FILE__ <<  ", line " << __LINE__
           << ", in " << __func__ << lw.endl;
        rslt_match.resize(1);
        rslt_match.at(0).coef = 0;
    }
    if(dm::log){
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_end);
        host_elapsed = clock_diff (&tp_start, &tp_end);
        clock_gettime(CLOCK_MONOTONIC, &tp_start);
        lw << "[DEBUG] Time elapsed - Result Computation: " << host_elapsed << "s" << lw.endl;
    }
    ////////////////////////// END: NCC Score //////////////////////////

    if(dm::log){
        hipDeviceSynchronize();
    }
    clock_gettime(CLOCK_MONOTONIC, &tp_end);
    host_elapsed = clock_diff (&start_all, &tp_end);
    t_execute = host_elapsed;

    lw << "ChamferInference Done: " << host_elapsed << "s" << lw.endl;
    // num of non zero
    short template_size = (template_width + template_width) / 2;
    float best_score = score_candidates_result[best_case];
    lw << "template_size : " << template_size << lw.endl;
    lw << "best_score : " << best_score << lw.endl;
    if(0.5 > chamfer_score_threshold_ || chamfer_score_threshold_ > 10){
        chamfer_score_threshold_ = 3;
    }
    if( 0.5 < rslt_match.at(0).coef && rslt_match.at(0).coef < 0.8
       && best_score < chamfer_score_threshold_){
        lw << "[WARNING] Chamfer matching score is high. But NCC score is low." << lw.endl;
    }
    else if(rslt_match.at(0).coef < 0.8){
        fail_msg = "[FAILED]";
        fail_msg += "Failure 1: NCC matching score is lower than 0.8. ";

        for(int j = 0; j < num_result_scores; j++){
            int template_scale_rot = score_candidates_result_index[j] / length_source_candidate;
            int template_scale = template_scale_rot / num_angles;
            int template_rot = template_scale_rot % num_angles;
            int source_target = score_candidates_result_index[j] % length_source_candidate;

            lw << "Index: " << score_candidates_result_index[j] << ", Score: " << score_candidates_result[j] << lw.endl;
            lw << "[ " << source_candidate_col[source_target] << ", " << source_candidate_row[source_target] << " ]" << lw.endl;
            lw << "Scale: " << template_scale << ", Rot: " << template_rot << lw.endl;

            if(ncc_results_vec.size() > j){
                lw << "Score(NCC): " << ncc_results_vec.at(j)<< lw.endl;
            }
        }

        if(score_candidates_result[0] >= max_score_orig){
            fail_msg += "Failure 2: Chamfer matching score is too low. The current image may not have the shapes of the templates. Image processing parameters between templates and current images may be different. ";
        }

        lw << "[FAILED] Detection failed. Score: " << rslt_match.at(0).coef << lw.endl;
        lw << fail_msg << lw.endl;
        host_elapsed = 0;
        t_execute = host_elapsed;
        rslt_match.clear();
    }

    // Save result if matching is succeeded.
    if(host_elapsed > 0 && rslt_match.size() > 0){
        // Jang 20221019
        // Handling failed case
        if(rslt_match.at(0).coef > 0.4){
            FILE *result_score=fopen((data_path + result_score_path).c_str(),"wb");
            for(int i = 0; i < 1; i++){
                MatchData& match_data = rslt_match.at(i);
                // Jang 20221020
                // Fixed wrong y-value problem.
                fprintf(result_score, "%2d:        (%4d,%4d);%12.1f%;%12.1f;%12.1f\n",
                        i+1, match_data.x, orig_src_height - match_data.y
                        , match_data.coef * 100.0, _minScale + _scaleInterval * (float)match_data.scale, match_data.angle);

            }
            fclose(result_score); result_score = NULL;
        }
        else {
            // Jang 20221024
            host_elapsed = 0;
            t_execute = host_elapsed;
            rslt_match.clear();
            lw << "[FAILED] Detection Failed. " << "File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
        }
    }

    if(dm::log){
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_end);
        host_elapsed = clock_diff (&tp_start, &tp_end);
        lw << "[DEBUG] Time elapsed - Saving results: " << host_elapsed << "s" << lw.endl;
    }
    return (float)(host_elapsed);
#endif
    return 0;
}

float CudaChamfer::LoadChamferData(){

    struct timespec start_all;//clock_t start_all = clock();
    clock_gettime(CLOCK_MONOTONIC, &start_all);
    struct timespec tp_end;
    double host_elapsed;

    const std::string path_temp = data_path + "/templates";
    const std::string path_temp_pyr = data_path + "/templates_pyr";
    lw << "data_path: " << data_path << lw.endl;

    orig_template_img_mat_.release();
    pyr_template_img_mat_.release();
    orig_template_img_mat_ = cv::imread(data_path + "0_orig_template_img_mat.png");
    pyr_template_img_mat_ = cv::imread(data_path + "0_pyr_template_img_mat.png");

    if(pyr_template_img_mat_.empty() || orig_template_img_mat_.empty()){
        lw << "[ERROR] Templates do not exist." << lw.endl;
        return 0;
    }

    // Get the matching parameters. Jimmy. 20220112
    std::string params_path = this->params_path_ + parameter_matching_file;
    if(!fs::exists(params_path_)){
        if(!fs::create_directories(params_path_)){
            lw << "[ERROR] The path is invalid. Path: " << params_path_ << lw.endl;
            return 0;
        }
    }
    // Jang 20220419
    LoadParamsIniFile(params_path);
    if(chamfer_method_ == 0){
        if(!LoadChamferTemplate(trained_template_pyr, angle_min_, angle_max_,
                                angle_interval_coarse_align_, path_temp_pyr)) return 0;
        if(!LoadChamferTemplate(trained_template, angle_min_, angle_max_,
                                angle_interval_fine_align_, path_temp)) return 0;
    }
    else{
        trained_template_vec_.resize(down_sampling_chamfer_);
        if(!LoadChamferTemplate2(trained_template_vec_, angle_min_, angle_max_,
                                 angle_interval_fine_align_, angle_interval_coarse_align_, path_temp)) return 0;

    }

//    return 0;

    clock_gettime(CLOCK_MONOTONIC, &tp_end);
    host_elapsed = clock_diff (&start_all, &tp_end);
    lw << "[Loading Templates] " << (double)(host_elapsed) << lw.endl;
    return (float)host_elapsed;
}

// Jang 20211207
bool CudaChamfer::IsChamferTemplateOk(){
    if(chamfer_template_score > 0) return true;
    else return false;
}

float CudaChamfer::ChamferTrain(cv::Mat template_img){
    chamfer_template_score = 0.0;
    lw << "[ChamferTrain start]" << lw.endl;

    struct timespec start_all;//clock_t start_all = clock();
    clock_gettime(CLOCK_MONOTONIC, &start_all);
    struct timespec tp_end;
    double host_elapsed;

    float execute_t = 0;

    const std::string path_temp = data_path + "/templates";
    const std::string path_temp_pyr = data_path + "/templates_pyr";
    if(fs::exists(path_temp)){
       fs::remove_all(path_temp);
    }
    if(fs::exists(path_temp_pyr)){
       fs::remove_all(path_temp_pyr);
    }

#ifdef DEBUG_MODE
    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
       << ", in " << __func__ << lw.endl;
#endif

    if (!fs::exists(data_path)) fs::create_directories(data_path);

    std::string params_path = this->params_path_ + parameter_matching_file;
    if(!fs::exists(params_path_)){
        if(!fs::create_directories(params_path_)){
            lw << "[ERROR] The path is invalid. Path: " << params_path_ << lw.endl;
            return 0;
        }
    }
    // Jang 20220419
    LoadParamsIniFile(params_path);

    orig_template_img_mat_ = template_img.clone();
#ifdef DEBUG_MODE
    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
       << ", in " << __func__ << lw.endl;
#endif
    // Down-sampling
    cv::Mat pyr_tmp_img;
    cv::pyrDown(template_img, pyr_tmp_img, cv::Size(template_img.cols/2, template_img.rows/2));    // become 1/2 size
    pyr_template_img_mat_ = pyr_tmp_img.clone();

    bool result = false;
    result = cv::imwrite(data_path + "0_orig_template_img_mat.png", orig_template_img_mat_);
    if (!result) {
        lw << "[ERROR] Failed to save templates." << lw.endl;
    }
    result = cv::imwrite(data_path + "0_pyr_template_img_mat.png", pyr_template_img_mat_);
    if (!result) {
        lw << "[ERROR] Failed to save pyr_templates." << lw.endl;
    }

    lw << data_path + "templates/0_orig_template_img_mat.png" << lw.endl;
    if (!orig_template_img_mat_.empty()) {
        lw << orig_template_img_mat_.rows << "-" << orig_template_img_mat_.cols<< lw.endl;
    } else {
        lw << "orig_template_img_mat_.empty()"<< lw.endl;
    }


#ifdef DEBUG_MODE
    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
       << ", in " << __func__ << lw.endl;
#endif


    chamfer_template_score = 1;

    // Jang 20220711
    if(chamfer_method_ == 0){
        // Old method
        if(trained_template_pyr.templates.size() > 0){
            FreeMemory(&trained_template_pyr.flat_template_non_zero_col_);
            FreeMemory(&trained_template_pyr.flat_template_non_zero_row_);
            FreeMemory(&trained_template_pyr.num_non_zeros);
            trained_template_pyr.non_zero_area_templates.clear();
            trained_template_pyr.templates.clear();
            trained_template_pyr.templates_gray.clear();
            trained_template_pyr.template_angle.clear();
            // Jang 20220330
            FreeMemory(&trained_template_pyr.flat_template_weight_);
            trained_template_pyr.template_weights.clear();
        }
        if(trained_template.templates.size() > 0){
            FreeMemory(&trained_template.flat_template_non_zero_col_);
            FreeMemory(&trained_template.flat_template_non_zero_row_);
            FreeMemory(&trained_template.num_non_zeros);
            trained_template.non_zero_area_templates.clear();
            trained_template.templates.clear();
            trained_template.templates_gray.clear();
            trained_template.template_angle.clear();
            // Jang 20220330
            FreeMemory(&trained_template.flat_template_weight_);
            trained_template.template_weights.clear();
        }
        // Generate the Non-zero mat for template
        // Downsized image is already blurred, so that blurring is no need to be adapted.
        // Also removing contours is not needed.
        image_proc->SkipBlurContour(true, true);
        createTempNonZeroMat(pyr_tmp_img, trained_template_pyr, angle_min_, angle_max_,
                             angle_interval_coarse_align_, (int)(skip_pixel_ / 2), path_temp_pyr);
        if(!trained_template_pyr.is_template_ok){
            chamfer_template_score = 0;
            return 0 ;
        }
        // Original template needs blurring
        // But not removing contours
        image_proc->SkipBlurContour(false, true);
        createTempNonZeroMat(template_img, trained_template, angle_min_, angle_max_,
                             angle_interval_fine_align_, skip_pixel_, path_temp);
        if(!trained_template.is_template_ok){
            chamfer_template_score = 0;
            return 0 ;
        }
    }
    else{
        // Jang 20220530
        // Mem allocation
        short scale = std::pow(2, down_sampling_chamfer_ - 1);
        int min_size = 20;
        int min_w_h = (template_img.cols < template_img.rows) ? template_img.cols : template_img.rows;
        if(scale * min_size > min_w_h){
            int target_scale = min_w_h / min_size;
            int count = 0;
            while(target_scale > 0){
                target_scale /= 2;
                count++;
            }
            down_sampling_chamfer_ = count;
        }

        ImageParams iparam;
        this->image_proc->LoadParams(iparam);
        down_sampling_chamfer_ = iparam.downsample_step + 1;

        trained_template_vec_.resize(down_sampling_chamfer_);
        image_proc->SkipBlurContour(false, false);
        if(chamfer_method_ == 1){
            CreateTempNonZeroMat2(template_img, trained_template_vec_,
                                 angle_min_, angle_max_,
                                 angle_interval_fine_align_, angle_interval_coarse_align_,
                                 skip_pixel_, path_temp);
        }
    }



    clock_gettime(CLOCK_MONOTONIC, &tp_end);
    host_elapsed = clock_diff (&start_all, &tp_end);
    lw << "[Training] " << (double)(host_elapsed) << lw.endl;

    execute_t = (float)host_elapsed;
    return execute_t ;
}


// Jang 20220608
void CudaChamfer::CreateSrcNonZeroMat2(cv::Mat orig_src,
                               std::vector<PreprocessedSourcePtr> &preprocessed_source,
                               short num_pyrmd,
                               int temp_w, int temp_h,
                               std::string path)
{
    /// num_pyrmd: the number of pyramids for resizing image. e.g.) if num_pyrmd = 2, images will be original, original / 2
    ///             default = 1, which means original size.
    ///
    assert(num_pyrmd > 0);  // The number includes the original size.

    std::string orig_path = path;
    cv::Mat src_img = orig_src;//cv::Mat src_img = orig_src.clone();
    cv::Mat src_contours;
    cv::Mat src_contours_inv;
    struct timespec tp_total, tp_start, tp_end;
    double host_elapsed;
    if(dm::log) {
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_total);
        clock_gettime(CLOCK_MONOTONIC, &tp_start);
        lw << "[DEBUG] Start - createSrcNonZeroMat" << lw.endl;
    }

    if(src_img.channels() > 1){
        cv::cvtColor(src_img, src_img, cv::COLOR_RGB2GRAY);
    }


    int src_w = orig_src.cols;
    int src_h = orig_src.rows;


    // Compute candidate with the smallest size
    // Detection will be precessed from the smallest image.

    for (int i = 0; i < preprocessed_source.size(); ++i) {
        if(use_only_start_and_end_){
            if(i != 0 && i != (preprocessed_source.size() - 1)){
                continue;
            }
        }

        path = orig_path + "/scale_" + std::to_string(i) + "/";
        fs::create_directories(path);
        PreprocessedSourcePtr& preproc_src = preprocessed_source.at(i);
        short resize_scale = std::pow(2, i); // 1, 2, 4, 8, ...
        int re_src_w = src_w / resize_scale;
        int re_src_h = src_h / resize_scale;
        int re_temp_w = temp_w / resize_scale;
        int re_temp_h = temp_h / resize_scale;
        cv::Size re_src_size = cv::Size(re_src_w, re_src_h);
        cv::Size re_temp_size = cv::Size(re_temp_w, re_temp_h);

        cv::Mat resized_img;
        cv::resize(src_img, resized_img, re_src_size);

        if(!use_image_proc) {
            cv::blur(resized_img, resized_img, cv::Size(3, 3));
            cv::Canny(resized_img, src_contours, 100, 200, 3);
        }
        else {
            ImageParams iparam;
            this->image_proc->LoadParams(iparam);
            if(i > 0){
                image_proc->AdjustParametersForDownsampling(iparam, iparam, i);
            }
            this->image_proc->GetImageBySavedInfo(resized_img, src_contours, iparam, use_cuda_for_improc);
        }
        if(dm::match_img)
        {
            cv::imwrite(path + std::to_string(src_contours.cols) + "_src_contours_0.png", src_contours);
        }

        if(dm::log) {
            hipDeviceSynchronize();
            clock_gettime(CLOCK_MONOTONIC, &tp_end);
            host_elapsed = clock_diff (&tp_start, &tp_end);
            clock_gettime(CLOCK_MONOTONIC, &tp_start);
            lw << "[DEBUG] Time elapsed - Contour Detection: " << host_elapsed << "s" << lw.endl;
        }
        ////////////////////////////////// Mask //////////////////////////////////
        cv::Mat src_mask;
        if(src_mask_exists_){
            if(orig_source_img_mat_.rows == source_img_mask_.rows && orig_source_img_mat_.cols == source_img_mask_.cols){
                if(i > 0) {
                    if(!source_img_mask_.empty()){
                        cv::resize(source_img_mask_, pyr_source_img_mask_, re_src_size);
                    }
                    src_mask = pyr_source_img_mask_;
                }else{
                    src_mask = source_img_mask_;
                }
                src_contours = src_contours - src_mask;
            }
            else{
                src_mask_exists_ = false;
            }
        }
        ////////////////////////////////// END: Mask //////////////////////////////////

        short padding_w = re_temp_w / 2;
        short padding_h = re_temp_h / 2;
        int expand_w = re_src_w;
        int expand_h = re_src_h;
        if(expand_img_){
            expand_w = re_src_w + padding_w * 2; //temp_w;
            expand_h = re_src_h + padding_h * 2; //temp_h;
        }
        cv::Size expand_size;
        expand_size.width = expand_w;
        expand_size.height = expand_h;
        int img_mem_size = expand_w * expand_h;

        ////////////////////////////////// CUDA: Set the CUDA memory block //////////////////////////////////
        int block_size = 32;    // max: 32
        dim3 threads_in_a_block(block_size, block_size);    // 16x16 threads
        int num_blocks_w = (int)((expand_w + block_size - 1) / block_size);
        int num_blocks_h = (int)((expand_h + block_size - 1) / block_size);
        dim3 num_blocks(num_blocks_w, num_blocks_h);


        if(i == preprocessed_source.size() - 1) {
            uchar* contours_data = (uchar*)src_contours.data;

            uchar* cuda_input_mat_data_8uc1 = nullptr;
            uchar* cuda_output_mat_data_8uc1 = nullptr;
            hipMalloc(&cuda_input_mat_data_8uc1, re_src_w * re_src_h * sizeof(uchar));
            hipMemcpy(cuda_input_mat_data_8uc1, contours_data,
                       re_src_w * re_src_h  * sizeof(uchar), hipMemcpyHostToDevice);
            hipMalloc(&cuda_output_mat_data_8uc1, img_mem_size * sizeof(uchar));

            if(dm::log) {
                lw << ", File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
            }
            DilateAndMoveMat<<<num_blocks, threads_in_a_block>>>(cuda_input_mat_data_8uc1, cuda_output_mat_data_8uc1,
                                                                 re_src_w, re_src_h,
                                                                 re_temp_w / 2, re_temp_h / 2,
                                                                 1, // No scaling
                                                                 padding_w, padding_h);
            if(dm::log) {
                lw << ", File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
            }
            hipError_t cuda_error = hipGetLastError();
            if(cuda_error != hipError_t::hipSuccess){
                lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
                   << ", File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
            }
            cv::Mat cuda_dilated_contours = cv::Mat::zeros(expand_size, CV_8UC1);
            std::shared_ptr<uchar[]> dilated_contours_smt_ptr(new uchar[img_mem_size]());
            uchar* dilated_contours_data = dilated_contours_smt_ptr.get();
            hipMemcpy(dilated_contours_data, cuda_output_mat_data_8uc1, (img_mem_size) * sizeof(uchar), hipMemcpyDeviceToHost);
            cuda_dilated_contours = cv::Mat(expand_size, CV_8UC1, (uchar*)dilated_contours_data);

            /// Mask noise in boundary
            cv::Rect mask_out_of_range_w(re_src_w + re_temp_w / 2, 0, re_temp_w / 2, expand_h);
            cv::Rect mask_out_of_range_h(0, re_src_h + re_temp_h / 2, expand_w, re_temp_h / 2);
            cv::add(cuda_dilated_contours(mask_out_of_range_w), -255, cuda_dilated_contours(mask_out_of_range_w));
            cv::add(cuda_dilated_contours(mask_out_of_range_h), -255, cuda_dilated_contours(mask_out_of_range_h));
            ///
            if(dm::log){
                std::cout << "------------- TEST ---------------" << std::endl;
                std::cout << "[DEBUG] func: " << __func__ << ", line " <<  std::endl;
            }
            FreeCudaMemory(&cuda_input_mat_data_8uc1);
            FreeCudaMemory(&cuda_output_mat_data_8uc1);

            if(dm::log) {
                lw << ", File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
            }

            if(dm::match_img)
            {
                cv::imwrite(path + std::to_string(src_contours.cols) + "_resized_contours.png", src_contours);
                cv::imwrite(path + std::to_string(cuda_dilated_contours.cols) + "_cuda_dilated_contours.png", cuda_dilated_contours);
            }
            ////////////////////////////////// END: Set the CUDA memory block //////////////////////////////////

            // Visualisation of the candidates
            if(dm::match_img)
            {
                try{
                    cv::Mat color_mask = cv::Mat::zeros(src_contours.size(), CV_8UC3);
                    cv::Mat color_dilated_contours_mask;
                    std::vector<cv::Point> non_zero_;
                    cv::findNonZero(src_contours, non_zero_);

                    // Change pixels' color
                    for(int p = 0; p < non_zero_.size(); p++){
                        cv::Vec3b& vec3b = color_mask.at<cv::Vec3b>(non_zero_.at(p).y, non_zero_.at(p).x) ;
                        vec3b[0] = 0;
                        vec3b[1] = 0;
                        vec3b[2] = 255;
                    }
                    cv::Rect crop_roi = cv::Rect(padding_w, padding_h, color_mask.cols, color_mask.rows);
                    cv::cvtColor(cuda_dilated_contours, color_dilated_contours_mask, cv::COLOR_GRAY2BGR);
                    color_mask.copyTo(color_dilated_contours_mask(crop_roi), src_contours.clone());
                    cv::imwrite(path + std::to_string(expand_w) + "_cuda_dilated_contours_masked.png", color_dilated_contours_mask);
                }
                catch(cv::Exception e){
                    lw << "File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                    std::string error_what = e.what();
                    lw << error_what << lw.endl;
                }catch(...){
                    lw << "Error. " << "File " <<  __FILE__ <<  ", line " << __LINE__
                       << ", in " << __func__ << lw.endl;
                }
            }

            if(dm::log) {
                lw << ", File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
            }



            cv::Mat candidate_mat = cuda_dilated_contours;// = cv::Mat::zeros(dilated_contours.size(), dilated_contours.type());



            if(dm::log) {
                hipDeviceSynchronize();
                clock_gettime(CLOCK_MONOTONIC, &tp_end);
                host_elapsed = clock_diff (&tp_start, &tp_end);
                clock_gettime(CLOCK_MONOTONIC, &tp_start);
                lw << "[DEBUG] Time elapsed - morphologyEx: " << host_elapsed << "s" << lw.endl;
            }

            std::vector<cv::Point> non_zero_area;
            cv::findNonZero(candidate_mat, non_zero_area);  // non_zero_area: coordinate(idx) of non-zero value

            // Set candidate coordinate
            // Non zero area
            preproc_src.length_source_candidate = non_zero_area.size();     // The number of non-zero pixels
            // Process 1d to 2d
            if(preproc_src.length_source_candidate > 0){
                preproc_src.source_num_candidate_width = (short)sqrt(preproc_src.length_source_candidate) + 1;
                preproc_src.source_num_candidate_height = preproc_src.source_num_candidate_width;

                // Allocate memory
                MakeSharedPtr(preproc_src.source_candidate_col, preproc_src.length_source_candidate);
                MakeSharedPtr(preproc_src.source_candidate_row, preproc_src.length_source_candidate);

                for(int i = 0; i < preproc_src.length_source_candidate; i ++){
                    preproc_src.source_candidate_col[i] = non_zero_area.at(i).x;
                    preproc_src.source_candidate_row[i] = non_zero_area.at(i).y;
                }
                non_zero_area.clear();
            }
            preproc_src.num_pyrmd = num_pyrmd;

            if(dm::log) {
                hipDeviceSynchronize();
                clock_gettime(CLOCK_MONOTONIC, &tp_end);
                host_elapsed = clock_diff (&tp_start, &tp_end);
                clock_gettime(CLOCK_MONOTONIC, &tp_start);
                lw << ", File " <<  __FILE__ <<  ", line " << __LINE__
                   << ", in " << __func__ << lw.endl;
                lw << "[DEBUG] Time elapsed - findNonZero: " << host_elapsed << "s" << lw.endl;
            }
        }

        // Resize the image to both up-left and down-right
        // Expanding size: template size
        cv::Mat expanded_candidate_mat;
        if(expand_img_){
            expanded_candidate_mat = cv::Mat::zeros(expand_size, src_contours.type());
            cv::Rect moving_contour_to(padding_w, padding_h, re_src_w, re_src_h);
            src_contours.copyTo(expanded_candidate_mat(moving_contour_to));
            src_contours.release();
            src_contours = expanded_candidate_mat.clone();
        }
        // # Distance Transform
        cv::threshold(src_contours, src_contours_inv, 127, 255, cv::THRESH_BINARY_INV);
        if(dm::match_img){
            cv::imwrite(path + std::to_string(resized_img.cols) + "_src.png", resized_img);
            cv::imwrite(path + std::to_string(src_contours.cols) + "_src_contours.png", src_contours);
            cv::imwrite(path + std::to_string(src_contours_inv.cols) + "_contour_source_img_inv.png", src_contours_inv);
        }
        if(dm::log) {
            hipDeviceSynchronize();
            clock_gettime(CLOCK_MONOTONIC, &tp_end);
            host_elapsed = clock_diff (&tp_start, &tp_end);
            clock_gettime(CLOCK_MONOTONIC, &tp_start);
            lw << "[DEBUG] Time elapsed - Others: " << host_elapsed << "s" << lw.endl;
        }

        ////////////////////////////////// Create Chamfer Image for source image //////////////////////////////////
        // Calculates the distance to the closest zero pixel for each pixel of the source image
        cv::distanceTransform(src_contours_inv, preproc_src.chamfer_img, cv::DIST_L2, CV_16S);
        preproc_src.flat_source_dist_transfrom = (float*)preproc_src.chamfer_img.data;
        preproc_src.width = src_contours_inv.cols;
        preproc_src.height = src_contours_inv.rows;

        if(dm::log) {
            hipDeviceSynchronize();
            clock_gettime(CLOCK_MONOTONIC, &tp_end);
            host_elapsed = clock_diff (&tp_start, &tp_end);
            clock_gettime(CLOCK_MONOTONIC, &tp_start);
            lw << "[DEBUG] Time elapsed - distanceTransform(CV_16S): " << host_elapsed << "s" << lw.endl;
        }

//        cv::Mat test_dist_trans;
//        cv::distanceTransform(src_contours_inv, test_dist_trans, cv::DIST_L2, CV_8U);
//        if(dm::log) {
//            hipDeviceSynchronize();
//            clock_gettime(CLOCK_MONOTONIC, &tp_end);
//            host_elapsed = clock_diff (&tp_start, &tp_end);
//            clock_gettime(CLOCK_MONOTONIC, &tp_start);
//            lw << "[DEBUG] Time elapsed - distanceTransform(CV_8U): " << host_elapsed << "s" << lw.endl;
//        }
//        cv::Mat test_dist_trans3;
//        cv::distanceTransform(src_contours_inv, test_dist_trans3, cv::DIST_L2, CV_32F);
//        if(dm::log) {
//            hipDeviceSynchronize();
//            clock_gettime(CLOCK_MONOTONIC, &tp_end);
//            host_elapsed = clock_diff (&tp_start, &tp_end);
//            clock_gettime(CLOCK_MONOTONIC, &tp_start);
//            lw << "[DEBUG] Time elapsed - distanceTransform(CV_32F): " << host_elapsed << "s" << lw.endl;
//        }
//        cv::Mat test_dist_trans2;
//        cv::distanceTransform(src_contours_inv, test_dist_trans2, cv::DIST_L2, CV_16S);
//        if(dm::log) {
//            hipDeviceSynchronize();
//            clock_gettime(CLOCK_MONOTONIC, &tp_end);
//            host_elapsed = clock_diff (&tp_start, &tp_end);
//            clock_gettime(CLOCK_MONOTONIC, &tp_start);
//            lw << "[DEBUG] Time elapsed - distanceTransform(CV_16S again): " << host_elapsed << "s" << lw.endl;
//        }
//        if(dm::match_img){
//            cv::imwrite(path + std::to_string(test_dist_trans.cols) + "_chamfer_img.png", preproc_src.chamfer_img);
//            cv::imwrite(path + std::to_string(test_dist_trans.cols) + "_test_dist_trans.png", test_dist_trans);
//            cv::imwrite(path + std::to_string(test_dist_trans.cols) + "_test_dist_trans2.png", test_dist_trans2);
//            cv::imwrite(path + std::to_string(test_dist_trans.cols) + "_test_dist_trans3.png", test_dist_trans3);
//        }
    }

    if(dm::log) {
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &tp_end);
        host_elapsed = clock_diff (&tp_total, &tp_end);
        lw << "[DEBUG] Time elapsed - TOTAL (CreateSrcNonZeroMat) : " << host_elapsed << "s" << lw.endl;
    }
}

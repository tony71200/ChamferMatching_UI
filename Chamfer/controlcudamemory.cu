#include "chamfermatch.h"

template<typename T>
bool CudaChamfer::FreeCudaMemory(T** cuda_pointer_){
    try{
        hipError_t cuda_error = hipFree(*cuda_pointer_);
        if(cuda_error != hipError_t::hipSuccess){
            lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
            throw(0);
        }
    }
    catch(int n_){
        std::cout << "[WARNING] Trying to free CUDA memory which is NULL" << std::endl;
        return false;
    }
    *cuda_pointer_  = NULL;
    return true;
}

template<typename T>
void CudaChamfer::FreeMemory(T** pointer_){
    try{
        if(*pointer_ != NULL)
            delete[] *pointer_;
    }
    catch(...){
        std::cout << "[WARNING] Trying to free memory which is NULL" << std::endl;
    }
    *pointer_  = NULL;
}


bool CudaChamfer::FreeCudaMemory(int** cuda_pointer_){
    return FreeCudaMemory<int>(cuda_pointer_);
    try{
        hipError_t cuda_error = hipFree(*cuda_pointer_);
        if(cuda_error != hipError_t::hipSuccess){
            lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
            throw(0);
        }
    }
    catch(int n_){
        std::cout << "[WARNING] Trying to free CUDA memory which is NULL" << std::endl;
        return false;
    }
    *cuda_pointer_  = NULL;
    return true;
}

bool CudaChamfer::FreeCudaMemory(short** cuda_pointer_){
    return FreeCudaMemory<short>(cuda_pointer_);
    try{
        hipError_t cuda_error = hipFree(*cuda_pointer_);
        if(cuda_error != hipError_t::hipSuccess){
            lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
            throw(0);
        }
    }
    catch(int n_){
        std::cout << "[WARNING] Trying to free CUDA memory which is NULL" << std::endl;
        return false;
    }
    *cuda_pointer_  = NULL;
    return true;
}

bool CudaChamfer::FreeCudaMemory(float** cuda_pointer_){
    return FreeCudaMemory<float>(cuda_pointer_);
    try{
        hipError_t cuda_error = hipFree(*cuda_pointer_);
        if(cuda_error != hipError_t::hipSuccess){
            lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
            throw(0);
        }
    }
    catch(int n_){
        std::cout << "[WARNING] Trying to free CUDA memory which is NULL" << std::endl;
        return false;
    }
    *cuda_pointer_  = NULL;
    return true;
}

bool CudaChamfer::FreeCudaMemory(double** cuda_pointer_){
    return FreeCudaMemory<double>(cuda_pointer_);
    try{
        hipError_t cuda_error = hipFree(*cuda_pointer_);
        if(cuda_error != hipError_t::hipSuccess){
            lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
            throw(0);
        }
    }
    catch(int n_){
        std::cout << "[WARNING] Trying to free CUDA memory which is NULL" << std::endl;
        return false;
    }
    *cuda_pointer_  = NULL;
    return true;
}

bool CudaChamfer::FreeCudaMemory(uchar** cuda_pointer_){
    return FreeCudaMemory<uchar>(cuda_pointer_);
    try{
        hipError_t cuda_error = hipFree(*cuda_pointer_);
        if(cuda_error != hipError_t::hipSuccess){
            lw << "[ERROR] CUDA error: " << hipGetErrorString(cuda_error)
               << ", File " <<  __FILE__ <<  ", line " << __LINE__
               << ", in " << __func__ << lw.endl;
            throw(0);
        }
    }
    catch(int n_){
        std::cout << "[WARNING] Trying to free CUDA memory which is NULL" << std::endl;
        return false;
    }
    *cuda_pointer_  = NULL;
    return true;
}

void CudaChamfer::FreeMemory(short** pointer_){
    FreeMemory<short>(pointer_); return;
    try{
        if(*pointer_ != NULL)
            delete[] *pointer_;
    }
    catch(...){
        std::cout << "[WARNING] Trying to free memory which is NULL" << std::endl;
    }
    *pointer_  = NULL;
}

void CudaChamfer::FreeMemory(int** pointer_){
    FreeMemory<int>(pointer_); return;
    try{
        if(*pointer_ != NULL)
            delete[] *pointer_;
    }
    catch(...){
        std::cout << "[WARNING] Trying to free memory which is NULL" << std::endl;
    }
    *pointer_  = NULL;
}

void CudaChamfer::FreeMemory(float** pointer_){
    FreeMemory<float>(pointer_); return;
    try{
        if(*pointer_ != NULL)
            delete[] *pointer_;
    }
    catch(...){
        std::cout << "[WARNING] Trying to free memory which is NULL" << std::endl;
    }
    *pointer_  = NULL;
}

void CudaChamfer::FreeMemory(double** pointer_){
    FreeMemory<double>(pointer_); return;
    try{
        if(*pointer_ != NULL)
            delete[] *pointer_;
    }
    catch(...){
        std::cout << "[WARNING] Trying to free memory which is NULL" << std::endl;
    }
    *pointer_  = NULL;
}

void CudaChamfer::FreeMemory(uchar** pointer_){
    FreeMemory<uchar>(pointer_); return;
    try{
        if(*pointer_ != NULL)
            delete[] *pointer_;
    }
    catch(...){
        std::cout << "[WARNING] Trying to free memory which is NULL" << std::endl;
    }
    *pointer_  = NULL;
}

